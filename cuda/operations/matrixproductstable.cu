#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <algorithm> 
#include <math.h>

const int CU_BLOCK_SIZE = 32;
#define GridSize(size) (size/CU_BLOCK_SIZE + 1)

//array - input(args)
//ans_array - output(rets)
//n, m - dimensions for each input matrix
//amount - amount of elements in each matrix
//maxn, maxm - maximum N and M from n and m accordingly
__global__
void matrixproduct(float** array, float** ans_array, int* n, int* m, int amount, int maxn, int maxm) {
	int Col = blockIdx.x*blockDim.x+threadIdx.x;
	int Row = blockIdx.y*blockDim.y+threadIdx.y;
	if(Col < maxn && Row < maxm) { //filter unneeded threads
		int numCColumns = maxn;
		int numCRows = maxm;
		//if ((Row < m[0]) && (Col < n[0])){
		//	ans_array[0][Row*numCColumns + Col] = array[0][Row*numCColumns + Col]; //first matrix >> answer
			//printf("%f\n", ans_array[0][Row*numCColumns + Col]);
		//}
		int numARows, numAColumns, numBRows, numBColumns;
		for (int i = 0; i < amount; i++){
			numARows = numCRows; // A - left matrix
			numAColumns = numCColumns;
			numBRows = m[i]; // B - right matrix
			numBColumns = n[i];
			numCRows = numARows; // C - result matrix
			numCColumns = numBColumns;
			if ((Row < numARows) && (Col < numBColumns)){
    			float Cvalue = 0.0f;
    			for (int k = 0; k < numAColumns; ++k){
    				Cvalue += ans_array[0][Row*numAColumns + k] * array[i][k * numBColumns + Col];
    				printf("Col is %i, Row is %i, K is %i, %f x %f is %f\n",Col, Row, k, ans_array[0][Row*numAColumns + k], array[i][k * numBColumns + Col], ans_array[0][Row*numAColumns + k] * array[i][k * numBColumns + Col]);
    			}
    			ans_array[0][Row*numCColumns + Col] = Cvalue;
    		}
		}
	}
}

void init(int N, int M, int K1, int K2){
	hipError_t err;
	hipSetDevice(0);

	float *host_array = new float[M];
	std::cout<<"INPUT IS "<<N<<" VECTORS:"<<std::endl;
	for (int i = 0; i < M; i++){
		host_array[i] = i;
		std::cout<<host_array[i]<<" ";
	}
	std::cout<<std::endl;

	float **host_ptr_array;

	host_ptr_array = (float**)malloc(N*sizeof(float*));

//GPU memory allocation	
	for (int i = 0; i < N; i++){
		hipMalloc((void**)&host_ptr_array[i], M * sizeof(float));
		err =
			hipMemcpy(host_ptr_array[i], host_array, M * sizeof(float), hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			std::cerr << "1Err is " << hipGetErrorString(err) << std::endl;
		}
	}

	float **dev_ptr_array;
	hipMalloc(&dev_ptr_array, N * sizeof(float*));
	err =
		hipMemcpy(dev_ptr_array, host_ptr_array, N * sizeof(float*), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cerr << "2Err is " << hipGetErrorString(err) << std::endl;
	}

//Dimensions allocation
	int *dim1_array = new int[N];
	int *dev_dim1_array;
	int *dim2_array = new int[N];
	int *dev_dim2_array;
	std::cout<<"DIMS ARE:"<<std::endl;
	for (int i = 0; i < N; i++){
		if (i % 2 == 0){
			dim1_array[i] = K1;
			dim2_array[i] = K2;
		} else{
			dim1_array[i] = K2;
			dim2_array[i] = K1;
		}
		std::cout<<dim1_array[i]<<" "<<dim2_array[i]<<std::endl;;
	}
	std::cout<<std::endl;
	hipMalloc(&dev_dim1_array, N * sizeof(int));
	hipMalloc(&dev_dim2_array, N * sizeof(int));
	err =
		hipMemcpy(dev_dim1_array, dim1_array, N * sizeof(int), hipMemcpyHostToDevice);	
	if (err != hipSuccess) {
		std::cerr << "WeightsErr is " << hipGetErrorString(err) << std::endl;
	}
	err =
		hipMemcpy(dev_dim2_array, dim2_array, N * sizeof(int), hipMemcpyHostToDevice);	
	if (err != hipSuccess) {
		std::cerr << "WeightsErr is " << hipGetErrorString(err) << std::endl;
	}

//Dimensions preprocessing
	int maxN = *std::max_element(dim1_array, dim1_array+N);
	int maxM = *std::max_element(dim2_array, dim2_array+N);
	//int ans_dim = maxN*maxM;
	int maxdim = std::max(maxN, maxM);
	int ans_dim = maxdim*maxdim;
	float *ans_array = new float[ans_dim];
	for (int i = 0; i < maxdim; i++)
		for (int j = 0; j < maxdim; j++)
			i == j ? ans_array[i*maxdim + j] = 1 : ans_array[i*maxdim + j] = 0;

	float **ans_ptr_array;
	ans_ptr_array = (float**)malloc(sizeof(float*));
	hipMalloc((void**)&ans_ptr_array[0], ans_dim * sizeof(float));
	err =
		hipMemcpy(ans_ptr_array[0], ans_array, ans_dim * sizeof(float), hipMemcpyHostToDevice);
		if (err != hipSuccess) {
		std::cerr << "1Err is " << hipGetErrorString(err) << std::endl;
	}
	float **dev_ans_ptr_array;
	hipMalloc(&dev_ans_ptr_array, sizeof(float*));
	err =
		hipMemcpy(dev_ans_ptr_array, ans_ptr_array, sizeof(float*), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cerr << "2Err is " << hipGetErrorString(err) << std::endl;
	}		

//Get results
	std::chrono::time_point<std::chrono::system_clock> start, end;
	start = std::chrono::system_clock::now();
//CUDA FUNCTION INVOKATION
	//weightedsum<<<GridSize(M), CU_BLOCK_SIZE>>>(dev_ptr_array, dev_ans_ptr_array, dev_weights_array, N, M);
	matrixproduct<<<dim3(GridSize(maxdim), GridSize(maxdim), 1), dim3(CU_BLOCK_SIZE, CU_BLOCK_SIZE, 1)>>>(dev_ptr_array, dev_ans_ptr_array, dev_dim1_array, dev_dim2_array, N, maxN, maxM);
	//matrixproduct<<<dim3(GridSize(std::sqrt(M)), GridSize(std::sqrt(M)), 1), dim3(CU_BLOCK_SIZE, CU_BLOCK_SIZE, 1)>>>(dev_ptr_array, dev_ans_ptr_array, dev_dim1_array, dev_dim2_array, N, maxN, maxM);
//CUDA FUNCTION END
	
	end = std::chrono::system_clock::now();
	int elapsed_seconds = std::chrono::duration_cast<std::chrono::microseconds>
                             (end-start).count();
    std::cout<<"COMPUTATION1 TIME IS: "<<elapsed_seconds<<std::endl;

//Copy results back
	err =
        hipMemcpy(ans_ptr_array, dev_ans_ptr_array, sizeof(float *), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		std::cerr << "6Err is " << hipGetErrorString(err) << std::endl;
	}

//Print ans
	err =
	hipMemcpy(ans_array, ans_ptr_array[0], ans_dim * sizeof(float), hipMemcpyDeviceToHost);
	if (err != hipSuccess)
		std::cerr << "7Err is " << hipGetErrorString(err) << std::endl;
	for (int i=0; i<ans_dim; i++){
		std::cout << ans_array[i] << " ";
		if (i > 0 & (i + 1) % maxN == 0) std::cout << std::endl;
	}
	std::cout << std::endl;

//Free memory
	for (int i = 0; i < N; i++){
		hipFree(&host_ptr_array[i]);
	}
	hipFree(&dev_ptr_array);
	hipFree(&dev_ans_ptr_array);
	free(host_ptr_array);
	hipDeviceReset();	
}

int main(int argc, char **argv){
	if (argc < 2){
		std::cout<<"Please enter N and M"<<std::endl;
		return 0;
	}
	int N = atoi(argv[1]);
	int M = atoi(argv[2]);
	int K1 = atoi(argv[3]);
	int K2 = atoi(argv[4]);
	init(N, M, K1, K2);
	return 0;
}