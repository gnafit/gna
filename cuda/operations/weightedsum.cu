#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#include "cuElementary.hh"

#include "cuda_config_vars.h" 

/*
* @brief Weighted sum of N vectors of length M into one
* @return \f$\sum w * x\f$ 
*
* @author Ilya Lebedev
* @date 2018
*/

template <typename T>
__global__ void weightedsum(T** array, T** ans_array, T* weights, int n, int m) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= m) return;
	ans_array[0][x] = array[0][x] * weights[0];
	for (int i = 1; i < n; i++){
		ans_array[0][x] += array[i][x] * weights[i];
	}
}


/*
* @brief Weighted sum of N vectors of length M into one
* @return \f$\sum w * x + k\f$ 
*
* @author Ilya Lebedev
* @date 2018
*/
template<typename T>
__global__ void weightedsumfill(T** array, T** ans_array, T* weights, T k, int n, int m) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= m) return;
	ans_array[0][x] = array[0][x] * weights[0] + k;
	for (int i = 1; i < n; i++){
		ans_array[0][x] += array[i][x] * weights[i];
	}
}

template<typename T>
void cuweightedsum(T** array, T** ans_array, T* weights, int n, int m) {
	weightedsum<<<m/CU_BLOCK_SIZE+1, CU_BLOCK_SIZE>>>(array, ans_array, weights, n, m);
	hipDeviceSynchronize();
}
