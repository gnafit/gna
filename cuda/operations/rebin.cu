#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#include "cuElementary.hh"

#include "cuda_config_vars.h" 
//const int CU_BLOCK_SIZE = 32;
#define GridSize(size) (size/CU_BLOCK_SIZE + 1)



template <typename T>
__global__
void rebin(T** args, T** ints, T** rets, size_t argsize, size_t retsize ) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	//int y = blockDim.y * blockIdx.y + threadIdx.y;
        rets[0][x] = 0;
	for(int i = 0; i < retsize;i++) {
		rets[0][x] += ints[0][argsize*x+i]*args[0][i]
	}
}

template<typename T>
void curebin(T** args, T** ints, T** rets, size_t argsize, size_t retsize) {
	rebin<<<argsize / CU_BLOCK_SIZE + 1, CU_BLOCK_SIZE>>>(T** args, T** ints, T** rets, size_t argsize, retsize);
	hipDeviceSynchronize();
}

template void curebin<double> (double** args, double** ints, double** rets, size_t argsize, size_t retsize);
