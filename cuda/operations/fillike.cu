#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#include "cuElementary.hh"
#include "cuda_config_vars.h"

__global__
void fillike(size_t val, double** ans_array, int n) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= n) return;
	ans_array[0][x] = val*1.0;
}

__global__
void fillike(size_t val, float** ans_array, int n) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= n) return;
	ans_array[0][x] = val*1.0;
}

void cufilllike(size_t val, float** ans_array, int n) {
	fillike<<<n/CU_BLOCK_SIZE+1, CU_BLOCK_SIZE>>> (val, ans_array, n);
	hipDeviceSynchronize();
}


void cufilllike(size_t val, double** ans_array, int n) {
	std::cout << "IMHERE!" <<std::endl <<std::endl;
	fillike<<<n/CU_BLOCK_SIZE+1, CU_BLOCK_SIZE>>> (val, ans_array, n);
	std::cout << "IMHERE!" <<std::endl <<std::endl;
	hipDeviceSynchronize();
}
