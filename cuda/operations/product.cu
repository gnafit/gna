#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#include "cuElementary.hh"
#include "cuda_config_vars.h" 

/*
* @brief Element-wise product of N vectors of length M into one
* @return \f$c, c_i = a_i * b_i, i=1..M\f$ 
*
* @author Ilya Lebedev
* @date 2018
*/
template<typename T>
__global__ void d_product(T** array, T** ans_array, unsigned int n, unsigned int m) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= m) return;
	ans_array[0][x] = array[0][x];
	for (int i = 1; i < n; i++){
		ans_array[0][x] *= array[i][x];
	}
}

template <typename T>
void cuproduct(T** array, T** ans_array, unsigned int n, unsigned int m) {
	cuproduct<<<m/CU_BLOCK_SIZE+1, CU_BLOCK_SIZE>>>(array, ans_array, n, m);
}

template void cuproduct<double>(double** array, double** ans_array, unsigned int n, unsigned int m); 
