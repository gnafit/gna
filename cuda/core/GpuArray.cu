#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <functional>
#include "GpuArray.hh"
#include "DataLocation.hh"

#define GridSize(size) (size/CU_BLOCK_SIZE + 1)



template <typename T>
__global__ void vecAdd(T* res, T* inA, T* inB, size_t n) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= n) return;
	res[x] = inA[x] + inB[x];
}

template <typename T>
__global__ void vecMinus(T* res, T* inA, T* inB, size_t n) {
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        if (x >= n) return;
        res[x] = inA[x] - inB[x];
}


template <typename T>
__global__ void vecMult(T* res, T* inA, T* inB, size_t n) {
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        if (x >= n) return;
        res[x] = inA[x] * inB[x];
}

template <typename T>
__global__ void vecMult(T* res, T* inA, T inB, size_t n) {
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        if (x >= n) return;
        res[x] = inA[x] * inB;
}



template <typename T>
__global__ void setByValueGPU(T* res, T val, size_t n) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= n) return;
	res[x] = val;
}

template <typename T>
__global__ void vecMinusUnar(T* arrPtr, size_t arrSize) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
        if (x >= arrSize) return;
        arrPtr[x] = -arrPtr[x];
}

template <typename T>
GpuArray<T>::GpuArray(T* inHostPtr) {
#ifdef CU_DEBUG
	std::cout << "GPUArray is created but not inited " << std::endl;
#endif
	hostPtr = inHostPtr;
	if(inHostPtr == nullptr)    dataLoc = DataLocation::NoData;
	else 			    dataLoc = DataLocation::Host;
	syncFlag = SyncFlag::Unsynchronized;
}

template <typename T>
GpuArray<T>::GpuArray(size_t inSize, T* inHostPtr) {
        hostPtr = inHostPtr;
	if(inHostPtr == nullptr)    dataLoc = DataLocation::NoData;
	syncFlag = SyncFlag::Unsynchronized;
#ifdef CU_DEBUG
	std::cout << "GPU Array is created by size (constructor)" << std::endl;
#endif
	hipError_t err;
	arrSize = inSize;
	size_t alloc_size = sizeof(T) * inSize;
	err = hipMalloc((void**)&devicePtr, alloc_size);
	if (err != hipSuccess) {
#ifdef CU_DEBUG
		printf("ERROR: unable to  allocate!\n");
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
#endif
		dataLoc = DataLocation::Crashed;
	} else {
		deviceMemAllocated = true;
		dataLoc = DataLocation::InitializedOnly;
	}
}

template <typename T>
DataLocation GpuArray<T>::Init(size_t inSize, T* inHostPtr) {
        hostPtr = inHostPtr;
	if(inHostPtr == nullptr)    dataLoc = DataLocation::NoData;
	syncFlag = SyncFlag::Unsynchronized;
#ifdef CU_DEBUG
        std::cout << "GPU Array is inited by size (Init)" << std::endl;
#endif
        hipError_t err;
        arrSize = inSize;
        size_t alloc_size = sizeof(T) * inSize;
        err = hipMalloc((void**)&devicePtr, alloc_size);
        if (err != hipSuccess) {
#ifdef CU_DEBUG
                printf("ERROR: unable to  allocate!\n");
                std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
#endif
                dataLoc = DataLocation::Crashed;
        } else {
		deviceMemAllocated = true; 
                dataLoc = DataLocation::InitializedOnly;
        }
	return dataLoc;
}

template <typename T>
GpuArray<T>::~GpuArray() {
	hipFree(devicePtr);
}

template <typename T>
DataLocation GpuArray<T>::setByHostArray(T* inHostArr) {
	hipError_t err;
	err = hipMemcpy(devicePtr, inHostArr, sizeof(T) * arrSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
#ifdef CU_DEBUG
		printf("ERROR: unable to copy memory H2D!\n");
		std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
#endif
		dataLoc = DataLocation::Crashed;
	} else {
		dataLoc = DataLocation::Device;
	}
	return dataLoc;
}

template <typename T>
DataLocation GpuArray<T>::setByDeviceArray(T* inDeviceArr) {
	hipError_t err;
	err = hipMemcpy(devicePtr, inDeviceArr, sizeof(T) * arrSize, hipMemcpyDeviceToDevice);
	if (err != hipSuccess) {
#ifdef CU_DEBUG
		printf("ERROR: unable to copy memory D2D!\n");
		std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
#endif
		dataLoc = DataLocation::Crashed;
	} else {
		dataLoc = DataLocation::Device;
	}
        return dataLoc;
}

template <typename T>
DataLocation GpuArray<T>::setByValue(T value) {
	setByValueGPU<T><<<GridSize(arrSize), CU_BLOCK_SIZE>>>(devicePtr, value, arrSize);
	dataLoc = DataLocation::Device;
        return dataLoc;
}

template <typename T>
DataLocation GpuArray<T>::getContentToCPU(T* dst) {
	hipError_t err;
	err = hipMemcpy(dst, devicePtr, sizeof(T) * arrSize, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
#ifdef CU_DEBUG
		printf("ERROR: unable to get array values to host!\n");
		std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
#endif
		dataLoc = DataLocation::Crashed;
	} else {
		dataLoc = DataLocation::Host;
	}
	return dataLoc;
}

template <typename T>
DataLocation GpuArray<T>::getContent(T* dst) {
	hipError_t err;
	err = hipMemcpy(dst, devicePtr, sizeof(T) * arrSize,
			 hipMemcpyDeviceToDevice);
	if (err != hipSuccess) {
#ifdef CU_DEBUG
		printf("ERROR: unable to get array values!\n");
		std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
#endif
		dataLoc = DataLocation::Crashed;
	} else {
		dataLoc = DataLocation::Device;
	}
	return dataLoc;
}

template <typename T> 
void GpuArray<T>::sync_H2D() {
#ifdef CU_DEBUG_3
    	printf("Sync to H2D\n");
#endif
	hipError_t err;
	if (dataLoc == DataLocation::NotInitialized) {
		err = hipMalloc((void**)&devicePtr, arrSize * sizeof(T));
        	if (err != hipSuccess) {
#ifdef CU_DEBUG
                	printf("ERROR: unable to  allocate!\n");
                	std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
#endif
                	dataLoc = DataLocation::Crashed;
			return;
        	}
		deviceMemAllocated = true;
	}
	err = hipMemcpy(devicePtr, hostPtr, sizeof(T) * arrSize,
                         hipMemcpyHostToDevice);
        if (err != hipSuccess) {
#ifdef CU_DEBUG
                printf("ERROR: unable to transfer data H2D!\n");
                std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
#endif
                dataLoc = DataLocation::Crashed;
		syncFlag = SyncFlag::SyncFailed;
        } else {
                dataLoc = DataLocation::Device;
                syncFlag = SyncFlag::Synchronized;
        }
}

template <typename T>
void GpuArray<T>::sync_D2H() {
#ifdef CU_DEBUG_3
	printf("Sync D2H\n");
#endif
        hipError_t err;
        err = hipMemcpy(hostPtr, devicePtr, sizeof(T) * arrSize,
                         hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
#ifdef CU_DEBUG
                printf("ERROR: unable to transfer data D2H!\n");
                std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
#endif
                dataLoc = DataLocation::Crashed;
		syncFlag = SyncFlag::SyncFailed;
        } else {
                dataLoc = DataLocation::Host;
		syncFlag = SyncFlag::Synchronized;
        }
}


template <typename T>
void GpuArray<T>::sync(DataLocation loc) {
/**
Copies the actual data to the loc location
*/
  if (dataLoc == loc || syncFlag == SyncFlag::Synchronized) {
#ifdef CU_DEBUG_2
    std::cerr << "Relevant data on "<< loc << "  -- no synchronization needed" << std::endl;
#endif
  } else if((dataLoc == DataLocation::Device && loc == DataLocation::Host)) {
    sync_D2H();
  } else if((dataLoc == DataLocation::Host && loc == DataLocation::Device)) {
    sync_H2D();
  } else if (dataLoc == DataLocation::NoData) {
    throw std::runtime_error("Data is not initialized");
  } else {
    syncFlag = SyncFlag::SyncFailed;
#ifdef CU_DEBUG_2
    std::cerr << "Cannot be synchronized! Smth wrong: current location state is <" << dataLoc << ">, new data location state is <" << loc << ">" << std::endl;
#endif
  }
}


template <typename T>
void GpuArray<T>::synchronize() {
/**
Makes data the same on GPU and CPU
*/
  if (dataLoc == DataLocation::Device) {
    sync(DataLocation::Host);
    syncFlag = SyncFlag::Synchronized;
#ifdef CU_DEBUG_3
    printf("Sync to GPU\n");
#endif
  } else if(dataLoc == DataLocation::Host) {
    sync(DataLocation::Device);
    syncFlag = SyncFlag::Synchronized;
#ifdef CU_DEBUG_3
    printf("Sync to CPU\n");
#endif
  } else {
#ifdef CU_DEBUG_2
    std::cerr << "Unable to sync data as current GPU memory state is " <<  dataLoc << std::endl;
#endif
    syncFlag = SyncFlag::SyncFailed;
  }
}


template <typename F>
GpuArray<F>& GpuArray<F>::operator+=(GpuArray<F> &rhs) {
	int smallest_size = arrSize;
	if (arrSize != rhs.getArraySize()) {
#ifdef CU_DEBUG
		if(arrSize > rhs.getArraySize()) { smallest_size = rhs.getArraySize(); }
		std::cerr << "ERROR: Sizes of lhs and rhs are different! The "
			     "result may be not valid!"
			  << std::endl;
#endif
	}
	vecAdd<F><<<GridSize(smallest_size), CU_BLOCK_SIZE>>>(devicePtr, devicePtr, rhs.getArrayPtr(),
				   smallest_size);
	dataLoc = DataLocation::Device;
	return *this;
}

template <typename F>
GpuArray<F>& GpuArray<F>::operator-=(GpuArray<F> &rhs) {
	int smallest_size = arrSize;
        if (arrSize != rhs.getArraySize()) {
                if(arrSize > rhs.getArraySize()) { smallest_size = rhs.getArraySize(); }
#ifdef CU_DEBUG
                std::cerr << "ERROR: Sizes of lhs and rhs are different! The "
                             "result may be not valid!"
                          << std::endl;
#endif
        }
        vecMinus<F><<<GridSize(smallest_size), CU_BLOCK_SIZE>>>(devicePtr, devicePtr, rhs.getArrayPtr(),
                                    smallest_size);
        dataLoc = DataLocation::Device;
        return *this;
}



template <typename F>
void GpuArray<F>::negate() {
	vecMinusUnar<F><<<GridSize(arrSize), CU_BLOCK_SIZE>>>(devicePtr, arrSize);
        dataLoc = DataLocation::Device;
}


template <typename F>
GpuArray<F>& GpuArray<F>::operator*=(GpuArray<F> &rhs) {
        size_t res_size = arrSize;
        if (arrSize != rhs.getArraySize()) {
		if(arrSize > rhs.getArraySize()) {res_size = rhs.getArraySize(); }
#ifdef CU_DEBUG 
                std::cerr << "ERROR: Sizes of lhs and rhs are different! The "
                             "result may be not valid!"
                          << std::endl;
#endif
        }
        vecMult<F><<<GridSize(res_size), CU_BLOCK_SIZE>>>(devicePtr, devicePtr, rhs.getArrayPtr(),
                                   res_size);
       dataLoc = DataLocation::Device;
       return *this; 
}


template <typename F>
GpuArray<F>& GpuArray<F>::operator*=(F rhs) {
        vecMult<F><<<GridSize(arrSize), CU_BLOCK_SIZE>>>(devicePtr, devicePtr, rhs,
                                   arrSize);
        dataLoc = DataLocation::Device;
        return *this;
}


template <typename T>
GpuArray<T> GpuArray<T>::operator=(GpuArray<T> rhs) {
	GpuArray<T> ret(rhs.arrSize);
	ret.setByDeviceArray(rhs.getArrayPtr());
        return ret;
}

template <typename T>
void GpuArray<T>::dump() {
	T* tmp = new T[arrSize];
	getContentToCPU(tmp);
	for (int i = 0; i < arrSize; i++) {
		std::cout << tmp[i] << " ";
        }
	std::cout << std::endl;
}

template class GpuArray<double>;
//template class GpuArray<float>;
//template class GpuArray<int>;
//template class GpuArray<bool>;
