#include "GpuBasics.hh"
#include "hip/hip_runtime.h"

#include <iostream>


template<typename T>
void device_malloc(T* &dst, unsigned int N) {
	hipMalloc(&dst, N*sizeof(T));
}

template<typename T>
void copyH2D_ALL(T* &dst, T* src, unsigned int N) {
	hipError_t err;
	hipMalloc(&dst, N * sizeof(T));
	err =
		hipMemcpy(dst, src, N * sizeof(T), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
	}
}

template<typename T>
void copyH2D_NA(T* dst, T* src, unsigned int N) {
        hipError_t err;
        err =
                hipMemcpy(dst, src, N * sizeof(T), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
                std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
        }
}



template<typename T>
void copyD2D_NA(T* dst, T* src, unsigned int N) {
        hipError_t err;
        err =
                hipMemcpy(dst, src, N * sizeof(T), hipMemcpyDeviceToDevice);
        if (err != hipSuccess) {
                std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
        }
}

template <typename T>
void cuwr_free(T* &ptr) {
	hipFree(ptr);
    ptr=nullptr;
}

template void copyH2D_ALL<unsigned int>(unsigned int* &dst, unsigned int* src, unsigned int N);
template void copyH2D_ALL<double>(double* &dst, double* src, unsigned int N);
template void cuwr_free<unsigned int>(unsigned int* &ptr);
template void cuwr_free<double>(double* &ptr);

template void copyH2D_ALL<unsigned int*>(unsigned int** &dst, unsigned int** src, unsigned int N);
template void copyH2D_ALL<double*>(double** &dst, double** src, unsigned int N);
template void cuwr_free<unsigned int*>(unsigned int** &ptr);
template void cuwr_free<double*>(double** &ptr);


template void copyH2D_NA<unsigned int*>(unsigned int** dst, unsigned int** src, unsigned int N);
template void copyH2D_NA<double*>(double** dst, double** src, unsigned int N);
template void copyH2D_NA<unsigned int>(unsigned int* dst, unsigned int* src, unsigned int N);
template void copyH2D_NA<double>(double* dst, double* src, unsigned int N);

template void device_malloc<double>(double* &dst, unsigned int N);
template void device_malloc<double*>(double** &dst, unsigned int N);
