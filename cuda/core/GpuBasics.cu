#include "GpuBasics.hh"
#include "hip/hip_runtime.h"

#include <iostream>

void copyH2D(double** dst, double** src, int N) {
	hipError_t err;
	hipMalloc(&dst, N * sizeof(double*));
	err =
		hipMemcpy(dst, src, N * sizeof(double*), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
	}
}
