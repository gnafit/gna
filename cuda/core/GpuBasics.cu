#include "GpuBasics.hh"
#include "hip/hip_runtime.h"
#include "cstddef"

#include <iostream>


template<typename T>
void debug_drop(T* in, unsigned int N) {
	T* tmp = new T[N];
	hipMemcpy(tmp, in, N * sizeof(T), hipMemcpyDeviceToHost);
	std::cout << "Debug drop:" << std::endl;
	for (unsigned int i = 0; i < N; ++i) {
		std::cout << tmp[i] << " ";
	}
	std::cout << std::endl;
}

template<typename T>
void debug_drop(T** in, size_t M /*how many arrs*/ , size_t N /*length of single arr*/ ) {
	std::cout << "Debug multidim drop:" << std::endl;
	T** bigtmp = new T*[M];
	hipMemcpy(bigtmp, in, M*sizeof(T*), hipMemcpyDeviceToHost);
	for(int j = 0; j < M ; j++) {
		T* tmp = new T[N];
		hipMemcpy(tmp, bigtmp[j], N * sizeof(T), hipMemcpyDeviceToHost);
		for (unsigned int i = 0; i < N; ++i) {
			std::cout << tmp[i] << " ";
		}
		std::cout << std::endl;
	}
	std::cout <<  std::endl;
}

template<typename T>
void device_malloc(T* &dst, unsigned int N) {
	hipError_t err = hipMalloc(&dst, N*sizeof(T));
	if (err != hipSuccess) {
		std::cerr << "Allocation err is " << hipGetErrorString(err) << std::endl;
	}
}

template<typename T>
void copyH2D_ALL(T* &dst, T* src, unsigned int N) {
	hipError_t err;
	hipMalloc(&dst, N * sizeof(T));
	err =
		hipMemcpy(dst, src, N * sizeof(T), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
	}
}

template<typename T>
void copyH2D_NA(T* dst, T* src, unsigned int N) {
        hipError_t err;
        err =
                hipMemcpy(dst, src, N * sizeof(T), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
                std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
        }
}



template<typename T>
void copyD2D_NA(T* dst, T* src, unsigned int N) {
        hipError_t err;
        err =
                hipMemcpy(dst, src, N * sizeof(T), hipMemcpyDeviceToDevice);
        if (err != hipSuccess) {
                std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
        }
}

template <typename T>
void cuwr_free(T* &ptr) {
	hipFree(ptr);
//    ptr=nullptr;
}

template void copyH2D_ALL<unsigned int>(unsigned int* &dst, unsigned int* src, unsigned int N);
template void copyH2D_ALL<size_t>(size_t* &dst, size_t* src, unsigned int N);
template void copyH2D_ALL<double>(double* &dst, double* src, unsigned int N);
template void cuwr_free<unsigned int>(unsigned int* &ptr);
template void cuwr_free<size_t>(size_t* &ptr);
template void cuwr_free<double>(double* &ptr);

template void copyH2D_ALL<unsigned int*>(unsigned int** &dst, unsigned int** src, unsigned int N);
template void copyH2D_ALL<size_t*>(size_t** &dst, size_t** src, unsigned int N);
template void copyH2D_ALL<double*>(double** &dst, double** src, unsigned int N);
template void cuwr_free<unsigned int*>(unsigned int** &ptr);
template void cuwr_free<size_t*>(size_t** &ptr);
template void cuwr_free<double*>(double** &ptr);


template void copyH2D_NA<unsigned int*>(unsigned int** dst, unsigned int** src, unsigned int N);
template void copyH2D_NA<double*>(double** dst, double** src, unsigned int N);
template void copyH2D_NA<unsigned int>(unsigned int* dst, unsigned int* src, unsigned int N);
template void copyH2D_NA<double>(double* dst, double* src, unsigned int N);

template void device_malloc<double>(double* &dst, unsigned int N);
template void device_malloc<double*>(double** &dst, unsigned int N);

template void debug_drop<double>(double* dst, unsigned int N);
template void debug_drop<double*>(double** dst, unsigned int N);
template void debug_drop<double>(double** in, size_t M /*how many arrs*/ , size_t N /*length of single arr*/ ) ;
