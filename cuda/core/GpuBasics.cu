#include "GpuBasics.hh"
#include "hip/hip_runtime.h"

#include <iostream>

template<typename T>
void copyH2D(T* dst, T* src, int N) {
	hipError_t err;
	hipMalloc(&dst, N * sizeof(T));
	err =
		hipMemcpy(dst, src, N * sizeof(T), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
	}
}

template<typename T>
void copyH2D_NOALL(T* dst, T* src, int N) {
        hipError_t err;
        err =
                hipMemcpy(dst, src, N * sizeof(T), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
                std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
        }
}

template <typename T>
void cuwr_free(T* ptr) {
	hipFree(ptr);
}

