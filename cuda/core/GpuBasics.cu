#include "GpuBasics.hh"
#include "hip/hip_runtime.h"

#include <iostream>

void copyH2D(double** dst, double** src, size_t N) {
	hipError_t err;
	err =
		hipMemcpy(dst, src, N * sizeof(double*), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
	}
}
