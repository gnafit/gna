#include "GpuBasics.hh"
#include "hip/hip_runtime.h"

#include <iostream>

void copyH2D(double** dst, double** src, int N) {
	hipError_t err;
	err =
		hipMemcpy(dst, src, N * sizeof(double*), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
	}
}
