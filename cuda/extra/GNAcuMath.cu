#include "hip/hip_runtime.h"
#define CUDA_API_PER_THREAD_DEFAULT_STREAM
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <typeinfo>
#include "GNAcuMath.h"
#include "hipblas.h"

/**
  *  Generation of Identity matrix on GPU memory
  */
__global__ void GenIdentity(int n, double* mat) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x < n && y < n) mat[x + n * y] = (x == y) ? 1.0 : 0.0;
}

/**
* cuBLAS multiplier wrapper for GNA
*/
void cuMultiplyMat(int m, int n, int k, double* InA, double* InB,
		   double* OutC) {
	hipSetDevice(0);
	hipblasHandle_t handle;
	hipblasStatus_t ret;
	hipError_t err;

	hipStream_t stream1;
	hipStreamCreate(&stream1);

	ret = hipblasCreate(&handle);
	if (ret != HIPBLAS_STATUS_SUCCESS) {
		printf("ERROR: unable to create cuBLAS handle!\n");
		exit(EXIT_FAILURE);
	}
	double* devA;
	double* devB;
	double* devC;
	hipMalloc((void**)&devA, m * k * sizeof(double));
	hipMalloc((void**)&devB, k * n * sizeof(double));
	hipMalloc((void**)&devC, m * n * sizeof(double));

	hipMemcpyAsync(devA, InA, m * k * sizeof(double),
			hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(devB, InB, k * n * sizeof(double),
			hipMemcpyHostToDevice, stream1);
	hipMemset(devC, 0, m * n * sizeof(double));
	double alpha = 1, beta = 0;
	hipDeviceSynchronize();
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, devA, m,
		    devB, k, &beta, devC, m);

	err = hipMemcpyAsync(OutC, devC, m * n * sizeof(double),
			      hipMemcpyDeviceToHost, stream1);

	if (err != hipSuccess) {
		printf("ERROR: unable to copy memory from device to host! \n");
		exit(EXIT_FAILURE);
	}
	hipStreamDestroy(stream1);
	hipblasDestroy(handle);
	hipFree(devA);
	hipFree(devB);
	hipFree(devC);
}

/**
* cuBLAS linear system solver wrapper for GNA. A is lower triangular.
*/
void cuSolveLowerLS(int m, int n, double* A, double* B) {
	hipblasHandle_t handle;
	hipblasStatus_t ret;
	hipError_t err;
	hipStream_t stream1;
	hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking);

	ret = hipblasCreate(&handle);
	if (ret != HIPBLAS_STATUS_SUCCESS) {
		printf("ERROR: unable to create cuBLAS handle!\n");
		exit(EXIT_FAILURE);
	}
	double* devA;
	double* devB;
	hipMalloc((void**)&devA, m * m * sizeof(double));
	hipMalloc((void**)&devB, m * n * sizeof(double));

	hipMemcpyAsync(devA, A, m * m * sizeof(double), hipMemcpyHostToDevice,
			stream1);
	hipMemcpyAsync(devB, B, m * n * sizeof(double), hipMemcpyHostToDevice,
			stream1);

	double alpha = 1.0;
	/**
	  *  Solve A*x =  B.
	  */
	ret = hipblasDtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
			     HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha,
			     devA, m, devB, m);

	hipDeviceSynchronize();
	if (ret != HIPBLAS_STATUS_SUCCESS) {
		printf("ERROR: unable to solve linear system with cuBLAS! \n");
		exit(EXIT_FAILURE);
	}

	err = hipMemcpyAsync(B, devB, m * n * sizeof(double),
			      hipMemcpyDeviceToHost, stream1);

	if (err != hipSuccess) {
		printf("ERROR: unable to copy memory from device to host! \n");
		exit(EXIT_FAILURE);
	}

	hipStreamDestroy(stream1);
	hipblasDestroy(handle);
	hipFree(devA);
	hipFree(devB);
}

/**
* cuBLAS matrix invertor wrapper for GNA. Uses cuBLAS linear system solver.
*/
void cuInverseMat(int matSize, double* InMat, double* OutMat) {
	const int blockSize = 16;
	int copyableSize = matSize * matSize * sizeof(double);
	hipSetDevice(0);
	hipblasHandle_t handle;
	hipblasStatus_t ret;
	hipError_t err;

	hipStream_t stream1, stream2;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);

	ret = hipblasCreate(&handle);
	if (ret != HIPBLAS_STATUS_SUCCESS) {
		printf("ERROR: unable to create cuBLAS handle!\n");
		exit(EXIT_FAILURE);
	}

	double* devInMat;
	double* devOutMat;
	hipMalloc((void**)&devInMat, copyableSize);
	hipMalloc((void**)&devOutMat, copyableSize);

	err = hipMemcpyAsync(devInMat, InMat, copyableSize,
			      hipMemcpyHostToDevice, stream1);
	if (err != hipSuccess) {
		printf("ERROR: unable to copy memory from host to device! \n");
		exit(EXIT_FAILURE);
	}

	GenIdentity<<<dim3(matSize / blockSize + 1, matSize / blockSize + 1),
		      dim3(blockSize, blockSize), 0, stream2>>>(matSize,
								devOutMat);

	double alpha = 1.0;
	/**
	  *  Solve A*x = alpha * B to invert matrix. In this case B is Identity,
	 * alpha == 1.
	  */
	ret =
	    hipblasDtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER,
			   HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, matSize, matSize,
			   &alpha, devInMat, matSize, devOutMat, matSize);

	hipDeviceSynchronize();
	if (ret != HIPBLAS_STATUS_SUCCESS) {
		printf("ERROR: unable to invert matrix with cuBLAS! \n");
		exit(EXIT_FAILURE);
	}

	err = hipMemcpyAsync(OutMat, devOutMat, copyableSize,
			      hipMemcpyDeviceToHost, stream1);

	if (err != hipSuccess) {
		printf("ERROR: unable to copy memory from device to host! \n");
		exit(EXIT_FAILURE);
	}
	hipStreamDestroy(stream1);
	hipStreamDestroy(stream2);
	hipblasDestroy(handle);
	hipFree(devInMat);
	hipFree(devOutMat);
	hipDeviceReset();
}
