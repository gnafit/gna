#include "hip/hip_runtime.h"
#include <iostream>
#include "cuInterpExpo.hh"
#include "cuda_config_vars.h"

/* Exp interpolation with segment indexes started with -1 in old way. 
 *
 */
__global__ void d_interpExpo(double** newx, double** newy, double* x, double* y, 
			int** xsegments, double* xwidths, int n) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	int cur_xsegm = xsegments[idx][idy];
	//int cur_xsegm = xsegments[idx];

	// check for underflow and overflow
	if (cur_xsegm < 0) {
		cur_xsegm = 0;
	}
	if (cur_xsegm >= n-1) { 
		cur_xsegm = n-2; 
	} 

	newy[idx][idy] = y[cur_xsegm] * exp( -(newx[idx][idy] - x[cur_xsegm]) *
	//newy[idx] = y[cur_xsegm] * exp( -(newx[idx] - x[cur_xsegm]) *
					log(y[cur_xsegm] / y[cur_xsegm+1]) /
					xwidths[cur_xsegm]
					);

	// NOTE: if we need to spend a lot of time for xwidths transfer
	//       we can avoid it by computing it as
	// 	 x[cur_xsegm+1] - x[cur_xsegm];
}


/* Exp interpolation with segment indexes started with 0 and without checking. 
 *
 */
__global__ void d_interpExpo(double** newx, double** newy, double* x, double* y, 
			int** xsegments, double* xwidths) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	int cur_xsegm = xsegments[idx][idy];

	newy[idx][idy] = y[cur_xsegm] * exp( -(newx[idx][idy] - x[cur_xsegm]) *
					log(y[cur_xsegm] / y[cur_xsegm+1]) /
					xwidths[cur_xsegm]
					);
}

__global__ void d_interpExpoA( double** args, double** rets, int Nold) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int cur_mat_idx = idx + Nold * idy;
	int cur_xsegm = args[2][cur_mat_idx];
	rets[idx][idy] = args[4][cur_xsegm] * 
			 exp ( -(args[0][cur_mat_idx] - args[1][cur_xsegm]) *
				log( args[4][cur_xsegm] / args[4][cur_xsegm + 1]) / 
				args[3][cur_xsegm] 
					); 
}




void interpExpo_v2(double** newx, double** newy, double* x, double* y,
			 int** xsegments, double* xwidths, int oldsize, int newsize) {

	d_interpExpo<<<dim3(newsize/CU_BLOCK_SIZE + 1, oldsize/CU_BLOCK_SIZE + 1), 
			dim3(CU_BLOCK_SIZE,CU_BLOCK_SIZE)>>>
			(newx, newy, x, y, xsegments, xwidths);
}


//void interpExpo_v1(double** newx, double** newy, double* x, double* y,
//			 int** xsegments, double* xwidths, int oldsize, int newsize) {

void interpExpo_v1(double** args, double** rets, int Nnew, int Nold) {
	d_interpExpoA<<<dim3(Nnew/CU_BLOCK_SIZE + 1, Nold/CU_BLOCK_SIZE + 1), 
			dim3(CU_BLOCK_SIZE,CU_BLOCK_SIZE)>>>
			(args, rets, Nold);
//	d_interpExpo<<<dim3(newsize/CU_BLOCK_SIZE + 1, oldsize/CU_BLOCK_SIZE + 1), 
//			dim3(CU_BLOCK_SIZE,CU_BLOCK_SIZE)>>>
//			(newx, newy, x, y, xsegments, xwidths, oldsize);
}

