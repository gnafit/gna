#include "GNAcuOscProbMem.hh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

template<typename T>
GNAcuOscProbMem<T>::GNAcuOscProbMem(int numOfElem) {
    hipError_t err;
    size_t alloc_size = sizeof(T) * numOfElem;
    hipSetDevice(0);

    err = hipMalloc((void**)&devEnu, alloc_size);
    if(err!=hipSuccess) {
      printf("ERROR: unable to  allocate!\n");
      std::cout << "err is " << hipGetErrorString(err) << std::endl;
    }
    err = hipMalloc((void**)&devTmp, alloc_size);
    if(err!=hipSuccess) {
      printf("ERROR: unable to  allocate!\n");
    }
    err = hipMalloc((void**)&devComp0, alloc_size);
    if(err!=hipSuccess) {
      printf("ERROR: unable to  allocate!\n");
    }
    err = hipMalloc((void**)&devComp12, alloc_size);
    if(err!=hipSuccess) {
      printf("ERROR: unable to  allocate!\n");
    }
    err = hipMalloc((void**)&devComp13, alloc_size);
    if(err!=hipSuccess) {
      printf("ERROR: unable to  allocate!\n");
    }
    err = hipMalloc((void**)&devComp23, alloc_size);
    if(err!=hipSuccess) {
      printf("ERROR: unable to  allocate!\n");
    }
    err = hipMalloc((void**)&devCompCP, alloc_size);
    if(err!=hipSuccess) {
      printf("ERROR: unable to  allocate!\n");
    }
    err = hipMalloc((void**)&devRet, alloc_size);
    if(err!=hipSuccess) {
      printf("ERROR: unable to  allocate!\n");
    }
}

template<typename T>
GNAcuOscProbMem<T>::~GNAcuOscProbMem() {
    hipFree(devComp0);
    hipFree(devCompCP);
    hipFree(devComp12);
    hipFree(devComp13);
    hipFree(devComp23);
    hipFree(devRet);
    hipFree(devTmp);
    hipFree(devEnu);
}

template class GNAcuOscProbMem<double>;
template class GNAcuOscProbMem<float>;

