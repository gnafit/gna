#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>



// tmp cuda programm -- block and grid size counter -- will be removed

__global__ void vecAdd(double* res, double* inA, double* inB, size_t n) {
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        if (x >= n) return;
        res[x] = inA[x] + inB[x];
}


int main () {
	int* minGridSize = (int*)malloc(sizeof(int));
	int* blockSize = (int*)malloc(sizeof(int));
	hipError_t err = hipOccupancyMaxPotentialBlockSize(minGridSize, blockSize, 
								vecAdd);
	std::cout << "minGridSize = " << *minGridSize << ", blockSize = " << *blockSize << std::endl;
	return 0;
}
