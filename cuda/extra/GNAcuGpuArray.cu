#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <functional>
#include "GNAcuGpuArray.hh"
#include "GNAcuDataLocation.hh"

template <typename T>
__global__ void vecAdd(T* res, T* inA, T* inB, size_t n) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= n) return;
	res[x] = inA[x] + inB[x];
}

template <typename T>
__global__ void vecMinus(T* res, T* inA, T* inB, size_t n) {
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        if (x >= n) return;
        res[x] = inA[x] - inB[x];
}


template <typename T>
__global__ void vecMult(T* res, T* inA, T* inB, size_t n) {
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        if (x >= n) return;
        res[x] = inA[x] * inB[x];
}

template <typename T>
__global__ void vecMult(T* res, T* inA, T inB, size_t n) {
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        if (x >= n) return;
        res[x] = inA[x] * inB;
}



template <typename T>
__global__ void setByValueGPU(T* res, T val, size_t n) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= n) return;
	res[x] = val;
}

template <typename T>
__global__ void vecMinusUnar(T* arrPtr, size_t arrSize) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
        if (x >= arrSize) return;
        arrPtr[x] = -arrPtr[x];
}

template <typename T>
GNAcuGpuArray<T>::GNAcuGpuArray() {
#ifdef CU_DEBUG
	std::cout << "GPUArray is created but not inited " << std::endl;
#endif
	arrState = NotInitialized;
}

template <typename T>
GNAcuGpuArray<T>::GNAcuGpuArray(size_t inSize) {
#ifdef CU_DEBUG
	std::cout << "GPU Array is created by size (constructor)" << std::endl;
#endif
	hipError_t err;
	arrSize = inSize;
	size_t alloc_size = sizeof(T) * inSize;
	err = hipMalloc((void**)&devicePtr, alloc_size);
/*
  std::chrono::time_point<std::chrono::system_clock> start, end;
  end = std::chrono::system_clock::now();
  std::time_t end_time = std::chrono::system_clock::to_time_t(end);
std::cout << "After Malloc Constructor: " << std::ctime(&end_time) << std::endl;
*/
	if (err != hipSuccess) {
#ifdef CU_DEBUG
		printf("ERROR: unable to  allocate!\n");
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
#endif
		arrState = Crashed;
	} else {
		arrState = InitializedOnly;
	}
}

template <typename T>
DataLocation GNAcuGpuArray<T>::Init(size_t inSize) {
#ifdef CU_DEBUG
        std::cout << "GPU Array is inited by size (Init)" << std::endl;
#endif
        hipError_t err;
        arrSize = inSize;
        size_t alloc_size = sizeof(T) * inSize;
        err = hipMalloc((void**)&devicePtr, alloc_size);
/*
std::chrono::time_point<std::chrono::system_clock> start, end;
  end = std::chrono::system_clock::now();
  std::time_t end_time = std::chrono::system_clock::to_time_t(end);
std::cout << "After Malloc Init: " << std::ctime(&end_time) << std::endl;
*/
        if (err != hipSuccess) {
#ifdef CU_DEBUG
                printf("ERROR: unable to  allocate!\n");
                std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
#endif
                arrState = Crashed;
        } else {
                arrState = InitializedOnly;
        }
	return arrState;
}

template <typename T>
GNAcuGpuArray<T>::~GNAcuGpuArray() {
	hipFree(devicePtr);
}

template <typename T>
void GNAcuGpuArray<T>::resize(size_t newSize) {
	hipError_t err;
	if (arrSize == newSize) return;
	if (arrSize > newSize) {
		arrSize = newSize;
		// TODO: free the end of array
		std::cerr << "WARNING! New array size is less then old size. "
			     "Some data may be lost!"
			  << std::endl;
	} else if (arrSize < newSize) {
		// TODO: resizing without realloc
		hipFree(devicePtr);
		size_t alloc_size = sizeof(T) * newSize;
		arrSize = newSize;
		err = hipMalloc((void**)&devicePtr, alloc_size);
                std::cout << "Resize: arrSize is " << arrSize << std::endl;

		if (err != hipSuccess) {
			printf("ERROR: unable to  allocate!\n");
			std::cerr << "err is " << hipGetErrorString(err)
				  << std::endl;
			arrState = Crashed;
		}
	}
}

template <typename T>
DataLocation GNAcuGpuArray<T>::setByHostArray(T* inHostArr) {
	hipError_t err;
	err = hipMemcpy(devicePtr, inHostArr, sizeof(T) * arrSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
#ifdef CU_DEBUG
		printf("ERROR: unable to copy memory H2D!\n");
		std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
#endif
		arrState = Crashed;
	} else {
		arrState = Device;
	}
	return arrState;
}

template <typename T>
DataLocation GNAcuGpuArray<T>::setByDeviceArray(T* inDeviceArr) {
	hipError_t err;
	err = hipMemcpy(devicePtr, inDeviceArr, sizeof(T) * arrSize, hipMemcpyDeviceToDevice);
	if (err != hipSuccess) {
#ifdef CU_DEBUG
		printf("ERROR: unable to copy memory D2D!\n");
		std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
#endif
		arrState = Crashed;
	} else {
		arrState = Device;
	}
        return arrState;
}

template <typename T>
DataLocation GNAcuGpuArray<T>::setByValue(T value) {
	setByValueGPU<T><<<arrSize, 1>>>(devicePtr, value, arrSize);
	arrState = Device;
        return arrState;
}

template <typename T>
DataLocation GNAcuGpuArray<T>::getContentToCPU(T* dst) {
	hipError_t err;
	err = hipMemcpy(dst, devicePtr, sizeof(T) * arrSize, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
#ifdef CU_DEBUG
		printf("ERROR: unable to get array values to host!\n");
		std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
#endif
		arrState = Crashed;
	} else {
		arrState = Host;
	}
	return arrState;
}

template <typename T>
DataLocation GNAcuGpuArray<T>::getContent(T* dst) {
	hipError_t err;
	err = hipMemcpy(dst, devicePtr, sizeof(T) * arrSize,
			 hipMemcpyDeviceToDevice);
	if (err != hipSuccess) {
#ifdef CU_DEBUG
		printf("ERROR: unable to get array values!\n");
		std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
#endif
		arrState = Crashed;
	} else {
		arrState = Device;
	}
	return arrState;
}

template <typename T> 
DataLocation GNAcuGpuArray<T>::transferH2D() {
	hipError_t err;
	if (arrState == NotInitialized) {
		err = hipMalloc((void**)&devicePtr, arrSize * sizeof(T));
        	if (err != hipSuccess) {
#ifdef CU_DEBUG
                	printf("ERROR: unable to  allocate!\n");
                	std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
#endif
                	arrState = Crashed;
        	}
	}
	err = hipMemcpy(devicePtr, hostPtr, sizeof(T) * arrSize,
                         hipMemcpyHostToDevice);
        if (err != hipSuccess) {
#ifdef CU_DEBUG
                printf("ERROR: unable to transfer data H2D!\n");
                std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
#endif
                arrState = Crashed;
        } else {
                arrState = Device;
        }
	return arrState;
}

template <typename T>
void GNAcuGpuArray<T>::transferD2H() {
        hipError_t err;
        hostPtr = new T[arrSize];
        err = hipMemcpy(hostPtr, devicePtr, sizeof(T) * arrSize,
                         hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
#ifdef CU_DEBUG
                printf("ERROR: unable to transfer data D2H!\n");
                std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
#endif
                arrState = Crashed;
        } else {
                arrState = Host;
        }
}


template <typename F>
GNAcuGpuArray<F>& GNAcuGpuArray<F>::operator+=(GNAcuGpuArray<F> &rhs) {
	int smallest_size = arrSize;
	if (arrSize != rhs.getArraySize()) {
#ifdef CU_DEBUG
		if(arrSize > rhs.getArraySize()) { smallest_size = rhs.getArraySize(); }
		std::cerr << "ERROR: Sizes of lhs and rhs are different! The "
			     "result may be not valid!"
			  << std::endl;
#endif
	}
	vecAdd<F><<<smallest_size, 1>>>(devicePtr, devicePtr, rhs.getArrayPtr(),
				   smallest_size);
	arrState = Device;
	return *this;
}

template <typename F>
GNAcuGpuArray<F>& GNAcuGpuArray<F>::operator-=(GNAcuGpuArray<F> &rhs) {
	int smallest_size = arrSize;
        if (arrSize != rhs.getArraySize()) {
                if(arrSize > rhs.getArraySize()) { smallest_size = rhs.getArraySize(); }
#ifdef CU_DEBUG
                std::cerr << "ERROR: Sizes of lhs and rhs are different! The "
                             "result may be not valid!"
                          << std::endl;
#endif
        }
        vecMinus<F><<<smallest_size, 1>>>(devicePtr, devicePtr, rhs.getArrayPtr(),
                                    smallest_size);
        arrState = Device;
        return *this;
}



template <typename F>
void GNAcuGpuArray<F>::negate() {
	vecMinusUnar<F><<<arrSize, 1>>>(devicePtr, arrSize);
        arrState = Device;
}


template <typename F>
GNAcuGpuArray<F>& GNAcuGpuArray<F>::operator*=(GNAcuGpuArray<F> &rhs) {
        size_t res_size = arrSize;
        if (arrSize != rhs.getArraySize()) {
		if(arrSize > rhs.getArraySize()) {res_size = rhs.getArraySize(); }
#ifdef CU_DEBUG 
                std::cerr << "ERROR: Sizes of lhs and rhs are different! The "
                             "result may be not valid!"
                          << std::endl;
#endif
        }
        vecMult<F><<<res_size, 1>>>(devicePtr, devicePtr, rhs.getArrayPtr(),
                                   res_size);
       arrState = Device;
       return *this; 
}


template <typename F>
GNAcuGpuArray<F>& GNAcuGpuArray<F>::operator*=(F rhs) {
        vecMult<F><<<arrSize, 1>>>(devicePtr, devicePtr, rhs,
                                   arrSize);
        arrState = Device;
        return *this;
}


template <typename T>
GNAcuGpuArray<T> GNAcuGpuArray<T>::operator=(GNAcuGpuArray<T> rhs) {
	GNAcuGpuArray<T> ret(rhs.arrSize);
	ret.setByDeviceArray(rhs.getArrayPtr());
        return ret;
}

template <typename T>
void GNAcuGpuArray<T>::dump() {
	T* tmp = new T[arrSize];
	getContentToCPU(tmp);
	for (int i = 0; i < arrSize; i++) {
		std::cout << tmp[i] << " ";
        }
	std::cout << std::endl;
}

template class GNAcuGpuArray<double>;
//template class GNAcuGpuArray<float>;
//template class GNAcuGpuArray<int>;
//template class GNAcuGpuArray<bool>;
