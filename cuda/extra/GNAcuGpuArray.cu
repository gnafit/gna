#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "GNAcuGpuArray.hh"
#include "GNAcuGpuMemStates.hh"

template <typename T>
__global__ void vecAdd(T* res, T* inA, T* inB, size_t n) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= n) return;
	res[x] = inA[x] + inB[x];
}

template <typename T>
__global__ void vecMinus(T* res, T* inA, T* inB, size_t n) {
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        if (x >= n) return;
        res[x] = inA[x] - inB[x];
}


template <typename T>
__global__ void vecMult(T* res, T* inA, T* inB, size_t n) {
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        if (x >= n) return;
        res[x] = inA[x] * inB[x];
}


template <typename T>
__global__ void setByValueGPU(T* res, T val, size_t n) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= n) return;
	res[x] = val;
}

template <typename T>
__global__ void vecMinusUnar(T* resPtr, T* arrayPtr, size_t arrSize) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
        if (x >= arrSize) return;
        resPtr[x] = -arrayPtr[x];
}

template <typename T>
GNAcuGpuArray<T>::GNAcuGpuArray() {
	std::cout << "I am created but not inited " << std::endl;
	arrState = NotInitialized;
}

template <typename T>
GNAcuGpuArray<T>::GNAcuGpuArray(T* inArrayPtr, size_t inSize) {
	std::cout << "I am created by ptr " << std::endl;
	hipError_t err;
	arrSize = inSize;
	size_t alloc_size = sizeof(T) * inSize;
	err = hipMalloc((void**)&arrayPtr, alloc_size);
	if (err != hipSuccess) {
		printf("ERROR: unable to  allocate!\n");
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
		arrState = Crashed;
	} else {
		arrState = InitializedOnly;
	}
}

template <typename T>
GNAcuGpuArray<T>::~GNAcuGpuArray() {
	hipFree(arrayPtr);
}

template <typename T>
void GNAcuGpuArray<T>::resize(size_t newSize) {
	hipError_t err;
	if (arrSize == newSize) return;
	if (arrSize > newSize) {
		arrSize = newSize;
		// TODO: free the end of array
		std::cerr << "WARNING! New array size is less then old size. "
			     "Some data may be lost!"
			  << std::endl;
	} else if (arrSize < newSize) {
		// TODO: resizing without realloc
		hipFree(arrayPtr);
		size_t alloc_size = sizeof(T) * newSize;
		arrSize = newSize;
		err = hipMalloc((void**)&arrayPtr, alloc_size);
		if (err != hipSuccess) {
			printf("ERROR: unable to  allocate!\n");
			std::cerr << "err is " << hipGetErrorString(err)
				  << std::endl;
			arrState = Crashed;
		}
	}
}

template <typename T>
void GNAcuGpuArray<T>::setByHostArray(T* inHostArr) {
	hipError_t err;
	err = hipMemcpy((void**)&arrayPtr, inHostArr, sizeof(T) * arrSize,
			 hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("ERROR: unable to set memory H2D!\n");
		std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
		arrState = Crashed;
	} else {
		arrState = OnDevice;
	}
}

template <typename T>
void GNAcuGpuArray<T>::setByDeviceArray(T* inDeviceArr) {
	hipError_t err;
	err = hipMemcpy(arrayPtr, inDeviceArr, sizeof(T) * arrSize,
			 hipMemcpyDeviceToDevice);
	if (err != hipSuccess) {
		printf("ERROR: unable to set memory D2D!\n");
		std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
		arrState = Crashed;
	} else {
		arrState = OnDevice;
	}
}

template <typename T>
void GNAcuGpuArray<T>::setByValue(T value) {
	setByValueGPU<T><<<arrSize, 1>>>(arrayPtr, value, arrSize);
	arrState = OnDevice;
}

template <typename T>
void GNAcuGpuArray<T>::getContentToCPU(T* dst) {
	hipError_t err;
	err = hipMemcpy(dst, arrayPtr, sizeof(T) * arrSize,
			 hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("ERROR: unable to get array values to host!\n");
		std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
		arrState = Crashed;
	} else {
		arrState = OnHost;
	}
}

template <typename T>
void GNAcuGpuArray<T>::getContent(T* dst) {
	hipError_t err;
	err = hipMemcpy(dst, arrayPtr, sizeof(T) * arrSize,
			 hipMemcpyDeviceToDevice);
	if (err != hipSuccess) {
		printf("ERROR: unable to get array values!\n");
		std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
		arrState = Crashed;
	} else {
		arrState = OnDevice;
	}
}

template <typename F>
GNAcuGpuArray<F> GNAcuGpuArray<F>::operator+(GNAcuGpuArray<F> rhs) {
	F* resPtr;
	size_t res_size = arrSize;
	if (arrSize != rhs.getArraySize()) {
		std::cerr << "ERROR: Sizes of lhs and rhs are different! The "
			     "smallest will be used!"
			  << std::endl;
		if (arrSize > rhs.getArraySize()) res_size = rhs.getArraySize();
	}
	hipError_t err;
	err = hipMalloc((void**)&resPtr, sizeof(F) * res_size);
	if (err != hipSuccess) {
		printf("ERROR: unable to  allocate memory for add result!\n");
		std::cerr << "err is " << hipGetErrorString(err) << std::endl;
	}
	vecAdd<F><<<res_size, 1>>>(resPtr, arrayPtr, rhs.getArrayPtr(),
				   res_size);
	F* ttt;
	GNAcuGpuArray<F> res(ttt, res_size);
	res.setByDeviceArray(resPtr);
	res.arrState = OnDevice;
	return res;
}

template <typename F>
GNAcuGpuArray<F> GNAcuGpuArray<F>::operator-(GNAcuGpuArray<F> rhs) {
        F* resPtr;
        size_t res_size = arrSize;
        if (arrSize != rhs.getArraySize()) {
                std::cerr << "ERROR: Sizes of lhs and rhs are different! The "
                             "smallest will be used!"
                          << std::endl;
                if (arrSize > rhs.getArraySize()) res_size = rhs.getArraySize();
        }
        hipError_t err;
        err = hipMalloc((void**)&resPtr, sizeof(F) * res_size);
        if (err != hipSuccess) {
                printf("ERROR: unable to  allocate memory for subtraction result!\n");
                std::cerr << "err is " << hipGetErrorString(err) << std::endl;
        }
        vecMinus<F><<<res_size, 1>>>(resPtr, arrayPtr, rhs.getArrayPtr(),
                                   res_size);
        F* ttt;
        GNAcuGpuArray<F> res(ttt, res_size);
        res.setByDeviceArray(resPtr);
        res.arrState = OnDevice;
        return res;
}



template <typename F>
GNAcuGpuArray<F> GNAcuGpuArray<F>::operator-() {
        F* resPtr;
        hipError_t err;
        err = hipMalloc((void**)&resPtr, sizeof(F) * arrSize);
	if (err != hipSuccess) {
                printf("ERROR: unable to  allocate memory for subtraction result!\n");
                std::cerr << "err is " << hipGetErrorString(err) << std::endl;
        }

	vecMinusUnar<F><<<arrSize, 1>>>(resPtr, arrayPtr, arrSize);
	F* ttt;
        GNAcuGpuArray<F> res(ttt, arrSize);
        res.setByDeviceArray(resPtr);
        res.arrState = OnDevice;
        return res;
}


template <typename F>
GNAcuGpuArray<F> GNAcuGpuArray<F>::operator*(GNAcuGpuArray<F> rhs) {
        F* resPtr;
        size_t res_size = arrSize;
        if (arrSize != rhs.getArraySize()) {
                std::cerr << "ERROR: Sizes of lhs and rhs are different! The "
                             "smallest will be used!"
                          << std::endl;
                if (arrSize > rhs.getArraySize()) res_size = rhs.getArraySize();
        }
        hipError_t err;
        err = hipMalloc((void**)&resPtr, sizeof(F) * res_size);
        if (err != hipSuccess) {
                printf("ERROR: unable to  allocate memory for add result!\n");
                std::cerr << "err is " << hipGetErrorString(err) << std::endl;
        }
        vecMult<F><<<res_size, 1>>>(resPtr, arrayPtr, rhs.getArrayPtr(),
                                   res_size);
        F* ttt;
        GNAcuGpuArray<F> res(ttt, res_size);
        res.setByDeviceArray(resPtr);
        res.arrState = OnDevice;
        return res;
}

template <typename T>
GNAcuGpuArray<T>& GNAcuGpuArray<T>::operator=(GNAcuGpuArray<T> rhs) {
	resize(rhs.getArraySize());
	(*this).arrState = OnDevice;
	(*this).setByDeviceArray(rhs.getArrayPtr());
	//(*this).arrState = OnDevice;
	return *this;
}

template class GNAcuGpuArray<double>;
template class GNAcuGpuArray<float>;
template class GNAcuGpuArray<int>;
template class GNAcuGpuArray<bool>;
