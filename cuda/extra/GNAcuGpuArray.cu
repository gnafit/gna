#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "GNAcuGpuArray.hh"
#include "GNAcuGpuMemStates.hh"

template <typename T>
__global__ void vecAdd(T* res, T* inA, T* inB, size_t n) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= n) return;
	res[x] = inA[x] + inB[x];
}

template <typename T>
__global__ void setByValueGPU(T* res, T val, size_t n) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= n) return;
	res[x] = val;
}

template <typename T>
GNAcuGpuArray<T>::GNAcuGpuArray() {
	std::cout << "I am created but not inited " << std::endl;
	arrState = NotInitialized;
}

template <typename T>
GNAcuGpuArray<T>::GNAcuGpuArray(T* inArrayPtr, size_t inSize) {
	std::cout << "I am created by ptr " << std::endl;
	hipSetDevice(0);
	hipError_t err;
	arrSize = inSize;
	size_t alloc_size = sizeof(T) * inSize;
	err = hipMalloc((void**)&arrayPtr, alloc_size);
	if (err != hipSuccess) {
		printf("ERROR: unable to  allocate!\n");
		std::cout << "err is " << hipGetErrorString(err) << std::endl;
		arrState = Crashed;
	} else {
		arrState = InitializedOnly;
	}
}

template <typename T>
GNAcuGpuArray<T>::~GNAcuGpuArray() {
	hipFree(arrayPtr);
	hipDeviceReset();
}

template <typename T>
void GNAcuGpuArray<T>::resize(size_t newSize) {
	hipError_t err;
	if (arrSize == newSize) return;
	if (arrSize > newSize) {
		arrSize = newSize;
		// TODO: free the end of array
		std::cerr << "WARNING! New array size is less then old size. "
			     "Some data may be lost!"
			  << std::endl;
	} else if (arrSize < newSize) {
		// TODO: resizing without realloc
		hipFree(arrayPtr);
		size_t alloc_size = sizeof(T) * newSize;
		arrSize = newSize;
		err = hipMalloc((void**)&arrayPtr, alloc_size);
		if (err != hipSuccess) {
			printf("ERROR: unable to  allocate!\n");
			std::cerr << "err is " << hipGetErrorString(err)
				  << std::endl;
			arrState = Crashed;
		}
	}
}

template <typename T>
void GNAcuGpuArray<T>::setByHostArray(T* inHostArr) {
	hipError_t err;
	err = hipMemcpy((void**)&arrayPtr, inHostArr, sizeof(T) * arrSize,
			 hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("ERROR: unable to set memory H2D!\n");
		std::cout << "Err is: " << hipGetErrorString(err) << std::endl;
		arrState = Crashed;
	} else {
		arrState = OnDevice;
	}
}

template <typename T>
void GNAcuGpuArray<T>::setByDeviceArray(T* inDeviceArr) {
	hipError_t err;
	err = hipMemcpy(arrayPtr, inDeviceArr, sizeof(T) * arrSize,
			 hipMemcpyDeviceToDevice);
	if (err != hipSuccess) {
		printf("ERROR: unable to set memory D2D!\n");
		std::cout << "Err is: " << hipGetErrorString(err) << std::endl;
		arrState = Crashed;
	} else {
		arrState = OnDevice;
	}
}

template <typename T>
void GNAcuGpuArray<T>::setByValue(T value) {
	setByValueGPU<T><<<arrSize, 1>>>(arrayPtr, value, arrSize);
	arrState = OnDevice;
}

template <typename T>
void GNAcuGpuArray<T>::getContentToCPU(T* dst) {
	hipError_t err;
	double* ttt = new double[arrSize];
	err = hipMemcpy(dst, arrayPtr, sizeof(T) * arrSize,
			 hipMemcpyDeviceToHost);

	if (err != hipSuccess) {
		printf("ERROR: unable to get array values to host!\n");
		std::cout << "Err is: " << hipGetErrorString(err) << std::endl;
		arrState = Crashed;
	} else {
		arrState = OnHost;
	}

	std::cout << std::endl;
}

template <typename T>
void GNAcuGpuArray<T>::getContent(T* dst) {
	hipError_t err;
	std::cout << "In getContent: arrSize = " << arrSize << std::endl;
	err = hipMemcpy(dst, arrayPtr, sizeof(T) * arrSize,
			 hipMemcpyDeviceToDevice);
	if (err != hipSuccess) {
		printf("ERROR: unable to get array values!\n");
		std::cout << "Err is: " << hipGetErrorString(err) << std::endl;
		arrState = Crashed;
	} else {
		arrState = OnDevice;
	}
}

template <typename F>
GNAcuGpuArray<F> GNAcuGpuArray<F>::operator+(GNAcuGpuArray<F> rhs) {
	F* resPtr;
	size_t res_size = arrSize;
	if (arrSize != rhs.getArraySize()) {
		std::cerr << "ERROR: Sizes of lhs and rhs are different! The "
			     "smallest will be used!"
			  << std::endl;
		if (arrSize > rhs.getArraySize()) res_size = rhs.getArraySize();
	}
	hipError_t err;
	err = hipMalloc((void**)&resPtr, sizeof(F) * res_size);
	if (err != hipSuccess) {
		printf("ERROR: unable to  allocate memory for add result!\n");
		std::cerr << "err is " << hipGetErrorString(err) << std::endl;
	}
	vecAdd<F><<<res_size, 1>>>(resPtr, arrayPtr, rhs.getArrayPtr(),
				   res_size);
	F* ttt;
	GNAcuGpuArray<F> res(ttt, res_size);
	res.setByDeviceArray(resPtr);
	res.arrState = OnDevice;
	return res;
}

template <typename T>
GNAcuGpuArray<T>& GNAcuGpuArray<T>::operator=(GNAcuGpuArray<T> rhs) {
// TODO rewrite
	hipError_t err;
	T* tmpArr;
	//`GNAcuGpuArray<T> res(tmpArr, rhs.getArraySize());
	err = hipMalloc((void**)&tmpArr, sizeof(T) * rhs.getArraySize());
	rhs.getContent(tmpArr);
	resize(rhs.getArraySize());
	//	arrayPtr = tmpArr;
	err = hipMemcpy(arrayPtr, tmpArr, sizeof(T) * rhs.getArraySize(),
			 hipMemcpyDeviceToDevice);

	// setByDeviceArray(tmpArr);

	std::cout << "in Op = : size = " << arrSize;
	//double* tmppp = new double[arrSize];
//	err = hipMemcpy(tmppp, arrayPtr, sizeof(T) * rhs.getArraySize(),
//			 hipMemcpyDeviceToHost);
/*	if (err != hipSuccess) {
		printf("ERROR: unable to  allocate!\n");
		std::cout << "err is " << hipGetErrorString(err) << std::endl;
		// res.arrState = Crashed;
	}
*/
/*
	getContentToCPU(tmppp);
	for (int i = 0; i < arrSize; i++) {
		std::cout << tmppp[i] << " ";
	}
	std::cout << std::endl;
*/
	return *this;
}

template class GNAcuGpuArray<double>;
template class GNAcuGpuArray<float>;
template class GNAcuGpuArray<int>;
template class GNAcuGpuArray<bool>;
