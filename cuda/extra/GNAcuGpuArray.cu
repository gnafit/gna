#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "GNAcuGpuArray.hh"
#include "GNAcuDataLocation.hh"

template <typename T>
__global__ void vecAdd(T* res, T* inA, T* inB, size_t n) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= n) return;
	res[x] = inA[x] + inB[x];
}

template <typename T>
__global__ void vecMinus(T* res, T* inA, T* inB, size_t n) {
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        if (x >= n) return;
        res[x] = inA[x] - inB[x];
}


template <typename T>
__global__ void vecMult(T* res, T* inA, T* inB, size_t n) {
        int x = blockDim.x * blockIdx.x + threadIdx.x;
        if (x >= n) return;
        res[x] = inA[x] * inB[x];
}


template <typename T>
__global__ void setByValueGPU(T* res, T val, size_t n) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= n) return;
	res[x] = val;
}

template <typename T>
__global__ void vecMinusUnar(T* resPtr, T* arrPtr, size_t arrSize) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
        if (x >= arrSize) return;
        resPtr[x] = -arrPtr[x];
}

template <typename T>
GNAcuGpuArray<T>::GNAcuGpuArray() {
	std::cout << "I am created but not inited " << std::endl;
	arrState = NotInitialized;
}

template <typename T>
GNAcuGpuArray<T>::GNAcuGpuArray(T* inArrayPtr, size_t inSize) {
	std::cout << "I am created by ptr " << std::endl;
	hipError_t err;
	arrSize = inSize;
	size_t alloc_size = sizeof(T) * inSize;
	err = hipMalloc((void**)&devicePtr, alloc_size);
	if (err != hipSuccess) {
		printf("ERROR: unable to  allocate!\n");
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
		arrState = Crashed;
	} else {
		arrState = InitializedOnly;
	}
}

template <typename T>
GNAcuGpuArray<T>::~GNAcuGpuArray() {
	hipFree(devicePtr);
}

template <typename T>
void GNAcuGpuArray<T>::resize(size_t newSize) {
	hipError_t err;
	if (arrSize == newSize) return;
	if (arrSize > newSize) {
		arrSize = newSize;
		// TODO: free the end of array
		std::cerr << "WARNING! New array size is less then old size. "
			     "Some data may be lost!"
			  << std::endl;
	} else if (arrSize < newSize) {
		// TODO: resizing without realloc
		hipFree(devicePtr);
		size_t alloc_size = sizeof(T) * newSize;
		arrSize = newSize;
		err = hipMalloc((void**)&devicePtr, alloc_size);
		if (err != hipSuccess) {
			printf("ERROR: unable to  allocate!\n");
			std::cerr << "err is " << hipGetErrorString(err)
				  << std::endl;
			arrState = Crashed;
		}
	}
}

template <typename T>
DataLocation GNAcuGpuArray<T>::setByHostArray(T* inHostArr) {
	hipError_t err;
	err = hipMemcpy((void**)&devicePtr, inHostArr, sizeof(T) * arrSize,
			 hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("ERROR: unable to set memory H2D!\n");
		std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
		arrState = Crashed;
	} else {
		arrState = Device;
	}
	return arrState;
}

template <typename T>
DataLocation GNAcuGpuArray<T>::setByDeviceArray(T* inDeviceArr) {
	hipError_t err;
	err = hipMemcpy(devicePtr, inDeviceArr, sizeof(T) * arrSize,
			 hipMemcpyDeviceToDevice);
	if (err != hipSuccess) {
		printf("ERROR: unable to set memory D2D!\n");
		std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
		arrState = Crashed;
	} else {
		arrState = Device;
	}
        return arrState;
}

template <typename T>
DataLocation GNAcuGpuArray<T>::setByValue(T value) {
	setByValueGPU<T><<<arrSize, 1>>>(devicePtr, value, arrSize);
	arrState = Device;
        return arrState;
}

template <typename T>
DataLocation GNAcuGpuArray<T>::getContentToCPU(T* dst) {
	hipError_t err;
	err = hipMemcpy(dst, devicePtr, sizeof(T) * arrSize,
			 hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("ERROR: unable to get array values to host!\n");
		std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
		arrState = Crashed;
	} else {
		arrState = Host;
	}
	return arrState;
}

template <typename T>
DataLocation GNAcuGpuArray<T>::getContent(T* dst) {
	hipError_t err;
	err = hipMemcpy(dst, devicePtr, sizeof(T) * arrSize,
			 hipMemcpyDeviceToDevice);
	if (err != hipSuccess) {
		printf("ERROR: unable to get array values!\n");
		std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
		arrState = Crashed;
	} else {
		arrState = Device;
	}
	return arrState;
}

template <typename T> 
DataLocation GNAcuGpuArray<T>::transferH2D() {
	hipError_t err;
	if (arrState == NotInitialized) {
		err = hipMalloc((void**)&devicePtr, arrSize * sizeof(T));
        	if (err != hipSuccess) {
                	printf("ERROR: unable to  allocate!\n");
                	std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
                	arrState = Crashed;
        	}
	}
	err = hipMemcpy(devicePtr, hostPtr, sizeof(T) * arrSize,
                         hipMemcpyHostToDevice);
        if (err != hipSuccess) {
                printf("ERROR: unable to transfer data H2D!\n");
                std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
                arrState = Crashed;
        } else {
                arrState = Device;
        }
	return arrState;
}

template <typename T>
DataLocation GNAcuGpuArray<T>::transferD2H() {
        hipError_t err;
        if (arrState == NotInitialized) {
                hostPtr = new T[arrSize];
        }
        err = hipMemcpy(hostPtr, devicePtr, sizeof(T) * arrSize,
                         hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
                printf("ERROR: unable to transfer data D2H!\n");
                std::cerr << "Err is: " << hipGetErrorString(err) << std::endl;
                arrState = Crashed;
        } else {
                arrState = Host;
        }
	return arrState;
}


template <typename F>
GNAcuGpuArray<F> GNAcuGpuArray<F>::operator+(GNAcuGpuArray<F> &rhs) {
	F* resPtr;
	size_t res_size = arrSize;
	if (arrSize != rhs.getArraySize()) {
		std::cerr << "ERROR: Sizes of lhs and rhs are different! The "
			     "smallest will be used!"
			  << std::endl;
		if (arrSize > rhs.getArraySize()) res_size = rhs.getArraySize();
	}
	hipError_t err;
	err = hipMalloc((void**)&resPtr, sizeof(F) * res_size);
	if (err != hipSuccess) {
		printf("ERROR: unable to  allocate memory for add result!\n");
		std::cerr << "err is " << hipGetErrorString(err) << std::endl;
	}
	vecAdd<F><<<res_size, 1>>>(resPtr, devicePtr, rhs.getArrayPtr(),
				   res_size);
	F* ttt = nullptr;
	GNAcuGpuArray<F> res(ttt, res_size);
	res.setByDeviceArray(resPtr);
	res.arrState = Device;
	return res;
}

template <typename F>
GNAcuGpuArray<F> GNAcuGpuArray<F>::operator-(GNAcuGpuArray<F> &rhs) {
        F* resPtr = nullptr;
        size_t res_size = arrSize;
        if (arrSize != rhs.getArraySize()) {
                std::cerr << "ERROR: Sizes of lhs and rhs are different! The "
                             "smallest will be used!"
                          << std::endl;
                if (arrSize > rhs.getArraySize()) res_size = rhs.getArraySize();
        }
        hipError_t err;
        err = hipMalloc((void**)&resPtr, sizeof(F) * res_size);
        if (err != hipSuccess) {
                printf("ERROR: unable to  allocate memory for subtraction result!\n");
                std::cerr << "err is " << hipGetErrorString(err) << std::endl;
        }
        vecMinus<F><<<res_size, 1>>>(resPtr, devicePtr, rhs.getArrayPtr(),
                                   res_size);
        F* ttt = nullptr;
        GNAcuGpuArray<F> res(ttt, res_size);
        res.setByDeviceArray(resPtr);
        res.arrState = Device;
        return res;
}



template <typename F>
GNAcuGpuArray<F> GNAcuGpuArray<F>::operator-() {
        F* resPtr = nullptr;
        hipError_t err;
        err = hipMalloc((void**)&resPtr, sizeof(F) * arrSize);
	if (err != hipSuccess) {
                printf("ERROR: unable to  allocate memory for subtraction result!\n");
                std::cerr << "err is " << hipGetErrorString(err) << std::endl;
        }

	vecMinusUnar<F><<<arrSize, 1>>>(resPtr, devicePtr, arrSize);
	F* ttt = nullptr;
        GNAcuGpuArray<F> res(ttt, arrSize);
        res.setByDeviceArray(resPtr);
        res.arrState = Device;
        return res;
}


template <typename F>
GNAcuGpuArray<F> GNAcuGpuArray<F>::operator*(GNAcuGpuArray<F> &rhs) {
        F* resPtr = nullptr;
        size_t res_size = arrSize;
        if (arrSize != rhs.getArraySize()) {
                std::cerr << "ERROR: Sizes of lhs and rhs are different! The "
                             "smallest will be used!"
                          << std::endl;
                if (arrSize > rhs.getArraySize()) res_size = rhs.getArraySize();
        }
        hipError_t err;
        err = hipMalloc((void**)&resPtr, sizeof(F) * res_size);
        if (err != hipSuccess) {
                printf("ERROR: unable to  allocate memory for add result!\n");
                std::cerr << "err is " << hipGetErrorString(err) << std::endl;
        }
        vecMult<F><<<res_size, 1>>>(resPtr, devicePtr, rhs.getArrayPtr(),
                                   res_size);
        F* ttt = nullptr;
        GNAcuGpuArray<F> res(ttt, res_size);
        res.setByDeviceArray(resPtr);
        res.arrState = Device;
        return res;
}

template <typename T>
GNAcuGpuArray<T>& GNAcuGpuArray<T>::operator=(GNAcuGpuArray<T> rhs) {
	resize(rhs.getArraySize());
	(*this).arrState = Device;
	(*this).setByDeviceArray(rhs.getArrayPtr());
	//(*this).arrState = Device;
	return *this;
}

template class GNAcuGpuArray<double>;
template class GNAcuGpuArray<float>;
template class GNAcuGpuArray<int>;
template class GNAcuGpuArray<bool>;
