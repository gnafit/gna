#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <typeinfo>

#include "cuRootMath.h"
#include "cuMathFunctions.h"
#include "cuOperations.cuh" 
#include <hip/hip_runtime.h>


template <int I, int J, typename T>
__global__ void cuCalcComponent(T* xarg, T* xret, T* intern); 
// TODO define inern as array of the same as xarg size from internals


template <>
__global__ void cuCalcComponent(float* xarg, float* xret, float* intern) { 
	inverse(xarg, intern);
	prodNumToVec (cosf(DeltaMSq<I,J>() * oscprobArgumentFactor * m_L * 0.5f), 
				intern,
				xret);	
}


template <>
__global__ void cuCalcComponent(double* xarg, double* xret, double* intern) { 
	inverse(xarg, intern);
	prodNumToVec (cosf(DeltaMSq<I,J>() * oscprobArgumentFactor * m_L * 0.5), 
				intern,
				xret);	
}


template <typename T>
__global__ void cuCalcComponentCP(T* xarg, T* xret, T* intern);

template <>
__global__ void cuCalcComponentCP(double* xarg, double* xret, double* intern) {
	inverse(xarg, intern);
	prodNumToVec ((oscprobArgumentFactor * m_L * 0.5), intern, intern);

	arr_sin(DeltaMSq<1,2>(), intern, xret);
	mult_by_arr_sin(DeltaMSq<1,3>(), intern, xret);
	mult_by_arr_sin(DeltaMSq<2,3>(), intern, xret);
}

