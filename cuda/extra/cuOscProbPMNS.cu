#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <typeinfo>

#include "cuRootMath.h"
#include "cuMathFunctions.h"
#include "cuOperations.cuh" 
#include <hip/hip_runtime.h>

/* TMP */

#define m_L 1.1
#define oscprobArgumentFactor 1.1

template<>
double OscProbPMNSBase::DeltaMSq<1,2>() const {return 0.1; }

template<>
double OscProbPMNSBase::DeltaMSq<1,3>() const {return 0.1; }

template<>
double OscProbPMNSBase::DeltaMSq<2,3>() const {return 0.1; }


/* TMP END */




template <int I, int J, typename T>
__global__ void cuCalcComponent(T* xarg, T* xret, T* intern); 
// TODO define inern as array of the same as xarg size from internals


template <int I, int J>
__global__ void cuCalcComponent(float* xarg, float* xret, float* intern) { 
	inverse(xarg, intern);
	prodNumToVec (cosf(DeltaMSq<I,J>() * oscprobArgumentFactor * m_L * 0.5f), 
				intern,
				xret);	
}


template <int I, int J>
__global__ void cuCalcComponent(double* xarg, double* xret, double* intern) { 
	inverse(xarg, intern);
	prodNumToVec (cosf(DeltaMSq<I,J>() * oscprobArgumentFactor * m_L * 0.5), 
				intern,
				xret);	
}

