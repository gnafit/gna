#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <typeinfo>

#include "cuRootMath.h"
#include "../operations/cuOperations.cuh" 
#include <hip/hip_runtime.h>
#include "cuda_config_vars.h"
#include "cuOscProbPMNS.hh"


/*
 * CUDA version of calcComponent function in OscProbPMNS::OscProbPMNS (floats)
 * 
 */

/*
template <int I, int J>
__global__ void cuCalcComponent(float** xarg, float** xret, float** intern) {
	inverse(xarg, intern);
	prodNumToVec (cosf(DeltaMSq<I,J>() * oscprobArgumentFactor * m_L * 0.5f), 
				intern,
				xret);	
}
*/

/*
 * CUDA version of calcComponent function in OscProbPMNS::OscProbPMNS (doubles)
 * 
 */

__global__ void d_cuCalcComponent(double** xarg, double** xret, double** intern, double** params,
				unsigned int m, double oscprobArgumentFactor, double DeltaMSq, double m_L) { 
	inverse(xarg[0], intern[0], m);
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
        xret[0][idx] = cos( DeltaMSq * m_L * oscprobArgumentFactor * 0.5 * intern[0][idx]);      
}


void cuCalcComponent(double** xarg, double** xret, double** intern, double** params,
		 unsigned int m, unsigned int n, double oscprobArgumentFactor, double DeltaMSq, double m_L) {
	d_cuCalcComponent<<<m/CU_BLOCK_SIZE + 1, CU_BLOCK_SIZE>>>(xarg, xret, intern, params,
								 m, oscprobArgumentFactor, DeltaMSq, m_L);
	hipDeviceSynchronize();
}

/*
 * CUDA version of calcComponentCP function in OscProbPMNS::OscProbPMNS
 * 
 */


//template <typename T>
__global__ void d_cuCalcComponentCP(double** xarg, double** xret, double** intern, double** params, unsigned int m,
					double oscprobArgumentFactor, double m_L) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	inverse(xarg[0], intern[0], m);
	intern[0][idx] = oscprobArgumentFactor * m_L * 0.25 * intern[0][idx];
	arr_sin(params[0][0], intern[0], xret[0]);
	mult_by_arr_sin(params[0][1], intern[0], xret[0]);
	mult_by_arr_sin(params[0][2], intern[0], xret[0]);
// for debug
//	arr_sin(1.0, intern, xret);
//	mult_by_arr_sin(1.0, intern, xret);
//	mult_by_arr_sin(1.0, intern, xret);
}

void cuCalcComponentCP(double** xarg, double** xret, double** intern, double** params, 
			unsigned int m, unsigned int n, double oscprobArgumentFactor, double m_L) {
	d_cuCalcComponentCP<<<m/CU_BLOCK_SIZE + 1, CU_BLOCK_SIZE>>>(xarg, xret, intern, params, m, 
								oscprobArgumentFactor, m_L);
	hipDeviceSynchronize();
}
