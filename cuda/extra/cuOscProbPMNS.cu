#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <typeinfo>

#include "cuRootMath.h"
#include "../operations/cuOperations.cuh" 
#include <hip/hip_runtime.h>


// TODO del debug values

//#define oscprobArgumentFactor 1.0
//#define m_L 1.0


// TODO define inern as array of the same as xarg size from internals


template <int I, int J>
__global__ void cuCalcComponent 
		(float* xarg, float* xret, float* intern) { 
	inverse(xarg, intern);
	prodNumToVec (cosf(DeltaMSq<I,J>() * oscprobArgumentFactor * m_L * 0.5f), 
				intern,
				xret);	
}


template < int I, int J>
__global__ void cuCalcComponent(double* xarg, double* xret, double* intern) { 
	inverse(xarg, intern);
	prodNumToVec (cos( DeltaMSq<I,J>() * oscprobArgumentFactor * m_L * 0.5), 
				intern,
				xret);	
}


template <typename T>
__global__ void cuCalcComponentCP(T* xarg, T* xret, T* intern) {
	inverse(xarg, intern);
	prodNumToVec ((oscprobArgumentFactor * m_L * T(0.5)), intern, intern);
	arr_sin(DeltaMSq<1,2>(), intern, xret);
	mult_by_arr_sin(DeltaMSq<1,3>(), intern, xret);
	mult_by_arr_sin(DeltaMSq<2,3>(), intern, xret);
// for debug
//	arr_sin(1.0, intern, xret);
//	mult_by_arr_sin(1.0, intern, xret);
//	mult_by_arr_sin(1.0, intern, xret);
}

