#include "hip/hip_runtime.h"
#include <cstdio>
#include <math.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include "simp.h"
#include "hipblas.h"
#define M 6
#define N 5
#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))

static __inline__ void modify (hipblasHandle_t handle, float *m, int ldm, int n, int p, int q, float alpha, float beta){
	    hipblasSscal (handle, n-p+1, &alpha, &m[IDX2F(p,q,ldm)], ldm);
	        hipblasSscal (handle, ldm-p+1, &beta, &m[IDX2F(p,q,ldm)], 1);
}


int notmain (void){
	hipError_t cudaStat;    
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	int i, j;
	float* devPtrA;
	float* a = 0;
	a = (float *)malloc (M * N * sizeof (*a));
	if (!a) {
		printf ("host memory allocation failed");
		return EXIT_FAILURE;
	}
	for (j = 1; j <= N; j++) {
		for (i = 1; i <= M; i++) {
			a[IDX2F(i,j,M)] = (float)((i-1) * M + j);
		}
	}
	cudaStat = hipMalloc ((void**)&devPtrA, M*N*sizeof(*a));
	if (cudaStat != hipSuccess) {
		printf ("device memory allocation failed");
		return EXIT_FAILURE;
	}
	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf ("CUBLAS initialization failed\n");
		return EXIT_FAILURE;
	}
	stat = hipblasSetMatrix (M, N, sizeof(*a), a, M, devPtrA, M);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf ("data download failed");
		hipFree (devPtrA);
		hipblasDestroy(handle);
		return EXIT_FAILURE;
	}
	modify (handle, devPtrA, M, N, 2, 3, 16.0f, 12.0f);
	stat = hipblasGetMatrix (M, N, sizeof(*a), devPtrA, M, a, M);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf ("data upload failed");
		hipFree (devPtrA);
		hipblasDestroy(handle);        
		return EXIT_FAILURE;
	}    
	hipFree (devPtrA);
	hipblasDestroy(handle);
	for (j = 1; j <= N; j++) {
		for (i = 1; i <= M; i++) {
			printf ("%7.0f", a[IDX2F(i,j,M)]);
		}
		printf ("\n");
	}
	free(a);
	return EXIT_SUCCESS;
}

const int NN = 50;

__global__ void  add( int *a,int *b, int *c ) {
int tid = blockIdx.x;    
if (tid < NN)
        c[tid] = a[tid] + b[tid];
}

void simpfun() {
    int  a[NN], b[NN], c[NN];
    int  *dev_a, *dev_b, *dev_c;
    hipSetDevice(0);
    hipMalloc( (void**)&dev_a, NN * sizeof(int ) );
    hipMalloc( (void**)&dev_b, NN * sizeof(int) );
    hipMalloc( (void**)&dev_c, NN * sizeof(int) );
    
    for (int i=0; i<NN; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    hipMemcpy( dev_a, a, NN * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, NN * sizeof(int), hipMemcpyHostToDevice );
    
    add<<<NN,1>>>( dev_a, dev_b, dev_c ); 
    
    hipMemcpy( c, dev_c, NN * sizeof(int), hipMemcpyDeviceToHost );
    for (int i=0; i<NN; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i] );
    }
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );
    int kkk = notmain();
    std::cout << "KKK is here: " << kkk << std::endl;
}
