#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <typeinfo>
#include <iostream>



void cuInverseMat(int matSize, double* InMat, double* OutMat) {
  hipblasHandle_t handle;
  hipblasStatus_t ret;
  hipError_t err;
  ret = hipblasCreate(&handle);
  if(ret!=HIPBLAS_STATUS_SUCCESS){
    printf("error code %d, line(%d)\n", ret, __LINE__);
    exit(EXIT_FAILURE);
  }

  double* devInMat;
  double* devOutMat;
  hipMalloc((void**)&devInMat,  matSize*matSize*sizeof(double));
  hipMalloc((void**)&devOutMat,  matSize*matSize*sizeof(double));

  err = hipMemcpy(devInMat, InMat, matSize*matSize*sizeof(double), hipMemcpyHostToDevice);
  if(err!=hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
  }

  err = hipMemcpy(devOutMat, OutMat, matSize*matSize*sizeof(double), hipMemcpyHostToDevice);
  if(err!=hipSuccess) {
    printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);
  }

  double alpha = 1.0;
  ret = hipblasDtrsm(handle,
          HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
          matSize, matSize, &alpha, devInMat, matSize, devOutMat, matSize);

  if(ret!=HIPBLAS_STATUS_SUCCESS) {
    printf("error code %d, line(%d)\n", ret, __LINE__);
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(OutMat, devOutMat, matSize*matSize*sizeof(double), hipMemcpyDeviceToHost);

  if(err!=hipSuccess) {
    printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);
  }

  printf("InMat:\n");
/*
  for (int i = 0; i < matSize; i++) {
    for (int j = 0; j < matSize; j++) {
      printf("%lf ", InMat[i + matSize*j]);
      
    }
    printf("\n");
  }
  printf("OutMat:\n");
  for (int i = 0; i < matSize; i++) {
    for (int j = 0; j < matSize; j++) {
      printf("%lf ", OutMat[i + matSize*j]);
    }
    printf("\n");
  }
*/
  hipFree(devInMat);
  hipFree(devOutMat);

}

int main() {
  const int matSize = 5000;
  double *InMat = new double[matSize*matSize];
  double *OutMat = new double[matSize*matSize];
  for(int i = 0; i < matSize; i++) {
    for (int j = 0; j < matSize; j++) {
      if (i == j){ OutMat [i+matSize*j] = 1.0;  InMat [i+matSize*j] = 2.0;}
      else {OutMat [i+matSize*j] = 0.0;  InMat [i+matSize*j] = 0.0;}
/*      if (!(j >= 3/2 && i < 3/2)) { 
         InMat[i+3*j] = 2.0;
      } 
      else { InMat[i+3*j] = 0.0; }
*/    }
  }
   printf("OutMat:\n");
  //int matSize = 3;
  /*for (int i = 0; i < matSize; i++) {
    for (int j = 0; j < matSize; j++) {
      printf("%lf ", OutMat[i + matSize*j]);
    }
    printf("\n");
  }
*/
  cuInverseMat(matSize, InMat, OutMat);
  return 0;
}

/*int cuInverseMatInPlace(int matSize, double* InvMat) {

}*/
