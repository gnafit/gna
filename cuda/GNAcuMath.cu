#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <typeinfo>
#include <iostream>

//#include "GNAcuMath.h"
#include "hip/hip_runtime_api.h"

/**
  *  Generation of Identity matrix on GPU memory
  *  TODO: find an optimal grid and block sizes!
  */
__global__ void GenIdentity (int n, double * mat) {
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  int y = blockDim.y*blockIdx.y + threadIdx.y;
  if (x < n && y < n)
  mat[x + n * y] = (x == y) ? 1.0 : 0.0; 
//   mat[x + n * y] = 15.0;
}

void cuInverseMat(int matSize, double* InMat, double* OutMat) {
//cudaProfilerStart();
  hipSetDevice(0);
  hipblasHandle_t handle;
  hipblasStatus_t ret;
  hipError_t err;
  ret = hipblasCreate(&handle);
  if(ret!=HIPBLAS_STATUS_SUCCESS){
    printf("error code %d, line(%d)\n", ret, __LINE__);
    exit(EXIT_FAILURE);
  }

  double* devInMat;
  double* devOutMat;
  hipMalloc((void**)&devInMat,  matSize*matSize*sizeof(double));
  hipMalloc((void**)&devOutMat,  matSize*matSize*sizeof(double));
//  cudaMallocManaged((void**)&devInMat,  matSize*matSize*sizeof(double));
//  cudaMallocManaged((void**)&devOutMat,  matSize*matSize*sizeof(double));
  GenIdentity<<<dim3(matSize/16 + 1, matSize/16 + 1), dim3(16,16)>>>(matSize, devOutMat);
  hipDeviceSynchronize();
  //err = cudaMemcpy(OutMat, devOutMat, matSize*matSize*sizeof(double), cudaMemcpyDeviceToHost);

  /*for (int i = 0; i < matSize; i++) {
    for (int j = 0; j < matSize; j++) {
      if (OutMat[i + j*matSize] != 0)
      std::cout << OutMat[i + j*matSize] << " ";
    }
    std::cout << std::endl;
  }
*/

  err = hipMemcpyAsync(devInMat, InMat, matSize*matSize*sizeof(double), hipMemcpyHostToDevice);
  if(err!=hipSuccess) {
#ifdef DEBUG
    printf("%s in %s at line %d\n", cudaGetErrorString(err), __FILE__, __LINE__);
#endif
    exit(EXIT_FAILURE);
  }

  double alpha = 1.0;
/**
  *  Solve A*x = alpha * B to invert matrix. In this case B is Identity, alpha == 1.
  */
  ret = hipblasDtrsm(handle,
          HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
          matSize, matSize, &alpha, devInMat, matSize, devOutMat, matSize);
  hipDeviceSynchronize();
  if(ret!=HIPBLAS_STATUS_SUCCESS) {
#ifdef  DEBUG
    printf("error code %d, line(%d)\n", ret, __LINE__);
#endif
    exit(EXIT_FAILURE);
  }

  err = hipMemcpyAsync(OutMat, devOutMat, matSize*matSize*sizeof(double), hipMemcpyDeviceToHost);

/*  if(err!=cudaSuccess) {
#ifdef DEBUG
    printf("%s in %s at line %d\n",cudaGetErrorString(err),__FILE__,__LINE__);
#endif
    exit(EXIT_FAILURE);
  }
*/
  hipFree(devInMat);
  hipFree(devOutMat);
//cudaProfilerStop();
}

int main () {
  int n = 200;
  double* inM = new double[n*n];
  double* outM = new double[n*n];
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      if (i == j) {
        inM[i + n*j] = i + j + 1;
        outM[i + n*j] = 1;
      }
      else {
        inM[i + n*j] = 0;
        outM[i + n*j] = 0;
      }
    }
  }

  for(int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      if (outM[i + j*n] != 0)
      std::cout << inM[i + j*n] << " ";
    }
  //  std::cout << std::endl;
  }

  cuInverseMat(n, inM,  outM);

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      if (outM[i + j*n] != 0) 
      std::cout << outM[i + j*n] << " ";
    }
    std::cout << std::endl;
  }

  return 0;
}

