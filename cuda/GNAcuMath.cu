#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <typeinfo>
#include <iostream>
#include <hip/hip_runtime.h>
#include "GNAcuMath.h"


/**
  *  Generation of Identity matrix on GPU memory
  *  TODO: find an optimal grid and block sizes!
  */
__global__ void GenIdentity (int n, double * mat) {
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  int y = blockDim.y*blockIdx.y + threadIdx.y;
  if (!(x >= n || x < 0 || y >= n || y < 0))  mat[x + n * y] = (x == y) ? 1.0 : 0.0;  
}

void cuInverseMat(int matSize, double* InMat, double* OutMat) {
  hipblasHandle_t handle;
  hipblasStatus_t ret;
  hipError_t err;
  ret = hipblasCreate(&handle);
  if(ret!=HIPBLAS_STATUS_SUCCESS){
    printf("error code %d, line(%d)\n", ret, __LINE__);
    exit(EXIT_FAILURE);
  }

  double* devInMat;
  double* devOutMat;
  hipMalloc((void**)&devInMat,  matSize*matSize*sizeof(double));
  hipMalloc((void**)&devOutMat,  matSize*matSize*sizeof(double));

  GenIdentity<<<1, dim3(matSize, matSize)>>>(matSize, devOutMat);
  
  err = hipMemcpy(devInMat, InMat, matSize*matSize*sizeof(double), hipMemcpyHostToDevice);
  if(err!=hipSuccess) {
#ifdef DEBUG
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
#endif
    exit(EXIT_FAILURE);
  }

  double alpha = 1.0;
  /**
  *  Solve A*x = alpha * B to invert matrix. In this case B is Identity, alpha == 1.
  */
  ret = hipblasDtrsm(handle,
          HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
          matSize, matSize, &alpha, devInMat, matSize, devOutMat, matSize);

  if(ret!=HIPBLAS_STATUS_SUCCESS) {
#ifdef  DEBUG
    printf("error code %d, line(%d)\n", ret, __LINE__);
#endif
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(OutMat, devOutMat, matSize*matSize*sizeof(double), hipMemcpyDeviceToHost);

  if(err!=hipSuccess) {
#ifdef DEBUG
    printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);
#endif
    exit(EXIT_FAILURE);
  }

  hipFree(devInMat);
  hipFree(devOutMat);
}

/*
int main() {
  const int matSize = 4096;
  double *InMat = new double[matSize*matSize];
  double *OutMat = new double[matSize*matSize];
  for(int i = 0; i < matSize; i++) {
    for (int j = 0; j < matSize; j++) {
      if (i == j){ OutMat [i+matSize*j] = 1.0;  InMat [i+matSize*j] = 2.0;}
      else {OutMat [i+matSize*j] = 0.0;  InMat [i+matSize*j] = 0.0;}
//      if (!(j >= 3/2 && i < 3/2)) { 
//         InMat[i+3*j] = 2.0;
//      } 
//      else { InMat[i+3*j] = 0.0; }
    }
  }
   printf("OutMat:\n");
  //int matSize = 3;
//  for (int i = 0; i < matSize; i++) {
//    for (int j = 0; j < matSize; j++) {
//      printf("%lf ", OutMat[i + matSize*j]);
//    }
//    printf("\n");
//  }

  cuInverseMat(matSize, InMat, OutMat);
  return 0;
}
*/
