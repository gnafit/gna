#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
//#include "cublas.h"
#include <typeinfo>
#include <iostream>



void cuInverseMat(int matSize, double* InMat, double* OutMat) {
  //if (typeid(InMat) != typeid(OutMat)) printf("Imput and output matrices types mismatch! Result can be wrong!"); 
/*  cudaError_t cudaStat;
  cublasStatus_t stat;
  cublasHandle_t handle;
  if (!InMat) {
     printf ("Input matrix is empty! Nothing to invert!");
     return EXIT_FAILURE;
  }
  cudaStat = cudaMalloc ((void**)&devInMat, matSize*matSize*sizeof(*InMat));
  if (cudaStat != cudaSuccess) {
    printf ("Device memory allocation failed with input matrix");
    return EXIT_FAILURE;
  }
  cudaStat = cudaMalloc ((void**)&devOutMat, matSize*matSize*sizeof(*OutMat));
  if (cudaStat != cudaSuccess) {
    printf ("Device memory allocation failed with output matrix");
    return EXIT_FAILURE;
  }
  stat = cublasSetMatrix(matSize, matSize, sizeof(InMat), InMat, M, devInMat, M); 
  if (stat != CUBLAS_STATUS_SUCCESS) {
    printf ("Input data download failed");
    cudaFree (devPtrA);
    cublasDestroy(handle);
    return EXIT_FAILURE;
  }*/
  hipblasHandle_t handle;
  hipblasStatus_t ret;
  hipError_t err;
  ret = hipblasCreate(&handle);
  if(ret!=HIPBLAS_STATUS_SUCCESS){
    printf("error code %d, line(%d)\n", ret, __LINE__);

switch (ret)
    {
        case HIPBLAS_STATUS_SUCCESS:
            printf("CUBLAS_STATUS_SUCCESS "); exit(EXIT_FAILURE);

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            printf("CUBLAS_STATUS_NOT_INITIALIZED" );exit(EXIT_FAILURE);

        case HIPBLAS_STATUS_ALLOC_FAILED:
            printf("CUBLAS_STATUS_ALLOC_FAILED ");exit(EXIT_FAILURE);

        case HIPBLAS_STATUS_INVALID_VALUE:
            printf("CUBLAS_STATUS_INVALID_VALUE ");exit(EXIT_FAILURE);

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            printf("CUBLAS_STATUS_ARCH_MISMATCH ");exit(EXIT_FAILURE);

        case HIPBLAS_STATUS_MAPPING_ERROR:
            printf("CUBLAS_STATUS_MAPPING_ERROR ");exit(EXIT_FAILURE);

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            printf("CUBLAS_STATUS_EXECUTION_FAILED ");exit(EXIT_FAILURE);

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            printf("CUBLAS_STATUS_INTERNAL_ERROR ");exit(EXIT_FAILURE);
    }


    exit(EXIT_FAILURE);
  }
  //UINT wTimerRes = 0;
 // bool init = InitMMTimer(wTimerRes);
 // startTime = timeGetTime();
  double* devInMat;
  double* devOutMat;
  hipMalloc((void**)&devInMat,  matSize*matSize*sizeof(*InMat));
  hipMalloc((void**)&devOutMat,  matSize*matSize*sizeof(*OutMat));
  err = hipMemcpy(devInMat, InMat, matSize*matSize*sizeof(*InMat), hipMemcpyHostToDevice);
  if(err!=hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
  }
  //err = cudaMemcpy(D_B,B,nCols*nCols*sizeof(float),_HTD);
  if(err!=hipSuccess) {
    printf("%s in %s at line %d\n",hipGetErrorString(err),__FILE__,__LINE__);
  }
//!!!!!!!!!
  const double alpha = 1.0;
  ret = hipblasDtrsm(handle,
          HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
          matSize, matSize, &alpha, devInMat, matSize, devOutMat, matSize);

  if(ret!=HIPBLAS_STATUS_SUCCESS) {
    printf("error code %d, line(%d)\n", ret, __LINE__);
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(OutMat, devOutMat, matSize*matSize*sizeof(*OutMat), hipMemcpyDeviceToHost);
  printf("InMat:\n");

  for (int i = 0; i < matSize; i++) {
    for (int j = 0; j < matSize; j++) {
      printf("%lf ", InMat[i + matSize*j]);
      
    }
    printf("\n");
  }
  printf("OutMat:\n");
  for (int i = 0; i < matSize; i++) {
    for (int j = 0; j < matSize; j++) {
      printf("%lf ", OutMat[i + matSize*j]);
    }
    printf("\n");
  }
  if(err!=hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
  }

  hipFree(devInMat);
  hipFree(devOutMat);

  //endTime = timeGetTime();
 // gtime=endTime-startTime;
  //std::cout << "GPU timing(including all device-host & host-device copies): " << float(gtime)/1000.0f << " seconds.\n";
  //DestroyMMTimer(wTimerRes, init);
}

int main() {
  double *InMat = new double[3*3];
  double *OutMat = new double[3*3];
  for(int i = 0; i < 3; i++) {
    for (int j = 0; j < 3; j++) {
      if (j <= 3/2 && i >= 3/2) InMat[i+3*j] = 0.0; 
      else InMat[i+3*j] = i+j+1; 
    }
  }
  cuInverseMat(3, InMat, OutMat);
  //return 0;
}

/*int cuInverseMatInPlace(int matSize, double* InvMat) {

}*/
