#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <typeinfo>
#include <iostream>
#include <hip/hip_runtime.h>
#include "GNAcuMath.h"

/**
  *  Generation of Identity matrix on GPU memory
  *  TODO: find an optimal grid and block sizes!
  */
__global__ void GenIdentity (int n, double * mat) {
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  int y = blockDim.y*blockIdx.y + threadIdx.y;
  if (x < n && y < n)
  mat[x + n * y] = (x == y) ? 1.0 : 0.0; 
}


/**
* cuBLAS multiplier wrapper for GNA 
*/
void cuMultiplyMat(int m, int n, int k, double* InA, double* InB, double* OutC) {
  hipSetDevice(0);
  hipblasHandle_t handle;
  hipblasStatus_t ret;
  hipError_t err;

  hipStream_t stream1, stream2;
  hipStreamCreate ( &stream1);
  hipStreamCreate ( &stream2);

  ret = hipblasCreate(&handle);
  if(ret!=HIPBLAS_STATUS_SUCCESS){
    exit(EXIT_FAILURE);
  }
  double* devA; double* devB; double* devC;
  hipMalloc((void**)&devA, m * k * sizeof(double));
  hipMalloc((void**)&devB, k * n * sizeof(double));
  hipMalloc((void**)&devC, m * n * sizeof(double));
  
  hipMemcpyAsync(devA, InA, m * k * sizeof(double), hipMemcpyHostToDevice, stream1);
  hipMemcpyAsync(devB, InB, k * n * sizeof(double), hipMemcpyHostToDevice, stream2);
  hipMemset(devC, 0, m * n * sizeof(double));
  double alpha = 1, beta = 0;
  hipDeviceSynchronize();  
  hipblasDgemm(handle, 
              HIPBLAS_OP_N,  HIPBLAS_OP_N,
              m, n, k,
              &alpha,
              devA, m,
              devB, k,
              &beta,
              devC, m );
  
  err = hipMemcpyAsync(OutC, devC, m * n * sizeof(double), hipMemcpyDeviceToHost, stream1);

  if(err!=hipSuccess) {
    exit(EXIT_FAILURE);
  }
  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipblasDestroy(handle);
  hipFree(devA);
  hipFree(devB);
  hipFree(devC);
}


/**
* cuBLAS linear system solver wrapper for GNA. A is lower triangular.
*/
void cuSolveLowerLS(int m, int n, double* A, double* B) {
//printf("BEF fffff \n");
//  hipSetDevice(0);
//printf("AF set dev \n");

  hipblasHandle_t handle;
  hipblasStatus_t ret;
  hipError_t err;
//printf("BEF str\n");
  hipStream_t stream1, stream2;
  hipStreamCreate ( &stream1);
  hipStreamCreate ( &stream2);

  ret = hipblasCreate(&handle);
  if(ret!=HIPBLAS_STATUS_SUCCESS){
    printf("hipblasCreate(&handle)");
    exit(EXIT_FAILURE);
  }
//printf("BEF cumalloc \n");
  double* devA;
  double* devB;
  hipMalloc((void**)&devA,  m*m*sizeof(double));
  hipMalloc((void**)&devB,  m*n*sizeof(double));
//printf("AF cumalloc \n");

  hipMemcpyAsync(devA, A, m*m*sizeof(double), hipMemcpyHostToDevice, stream1);
  hipMemcpyAsync(devB, B, m*n*sizeof(double), hipMemcpyHostToDevice, stream2);
  
  double alpha = 1.0;
/**
  *  Solve A*x =  B.
  */
  ret = hipblasDtrsm(handle,
          HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
          m, n, &alpha, devA, m, devB, m);


  hipDeviceSynchronize();
  if(ret!=HIPBLAS_STATUS_SUCCESS) {
    printf("hipblasDtrsm");
    exit(EXIT_FAILURE);
  }
  
  err = hipMemcpyAsync(B, devB, m*n*sizeof(double), hipMemcpyDeviceToHost, stream1);

  if(err!=hipSuccess) {
    printf("cudaMemcpyAsync0");
    exit(EXIT_FAILURE);
  }
  
  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2); 
  hipblasDestroy(handle);
  hipFree(devA);
  hipFree(devB);
}


/**
* cuBLAS matrix invertor wrapper for GNA. Uses cuBLAS linear system solver.
*/
void cuInverseMat(int matSize, double* InMat, double* OutMat) {
  const int blockSize = 16;
  int copyableSize = matSize*matSize*sizeof(double);
  hipSetDevice(0);
  hipblasHandle_t handle;
  hipblasStatus_t ret;
  hipError_t err;
  
  hipStream_t stream1, stream2;
  hipStreamCreate ( &stream1);
  hipStreamCreate ( &stream2);

  ret = hipblasCreate(&handle);
  if(ret!=HIPBLAS_STATUS_SUCCESS){
    exit(EXIT_FAILURE);
  }

  double* devInMat;
  double* devOutMat;
  hipMalloc((void**)&devInMat,  copyableSize);
  hipMalloc((void**)&devOutMat,  copyableSize);

  err = hipMemcpyAsync(devInMat, InMat, copyableSize, hipMemcpyHostToDevice, stream1);
  if(err!=hipSuccess) {
    exit(EXIT_FAILURE);
  }

  GenIdentity<<<dim3(matSize/blockSize + 1, matSize/blockSize + 1), dim3(blockSize, blockSize), 0, stream2>>>(matSize, devOutMat);

  double alpha = 1.0;
/**
  *  Solve A*x = alpha * B to invert matrix. In this case B is Identity, alpha == 1.
  */
  ret = hipblasDtrsm(handle,
          HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
          matSize, matSize, &alpha, devInMat, matSize, devOutMat, matSize);


  hipDeviceSynchronize();
  if(ret!=HIPBLAS_STATUS_SUCCESS) {
    exit(EXIT_FAILURE);
  }

  err = hipMemcpyAsync(OutMat, devOutMat, copyableSize, hipMemcpyDeviceToHost, stream1);

  if(err!=hipSuccess) {
    exit(EXIT_FAILURE);
  }
  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipblasDestroy(handle);
  hipFree(devInMat);
  hipFree(devOutMat);
}

/*int main () {
  int m = 20, n = 25;
  double* A = new double[m*m];
  //double* B = new double[k*n];
std::cout << "!!!!!!!!!!!!!!!!!!!!" << std::endl;
  double* C = new double[m*n];
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      if (i == j) {
        C[i + m*j] = i + j + 1;
        //C[i + n*j] = 1;
      }
      else {
       // if (j < n) A[i + n*j] = 0;
        C[i + m*j] = 0;
      }
    }
  }

  for(int i = 0; i < m; i++) {
    for (int j = 0; j < m; j++) {
      if (i == j) A[i + j*m] = 1; else A[i + j*m] = 0;
      std::cout << A[i + j*m] << " ";
    }
    std::cout << std::endl;
  }
std::cout << "BEFORE" <<std::endl;
  cuSolveLowerLS(m, n, A, C);
  //cuInverseMat(n, inM,  outM);
std::cout << "AFTER" << std::endl;
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      std::cout << C[i + j*m] << " ";
    }
    std::cout << std::endl;
  }
  return 0; 
}
*/
/*  for (int i = 0; i < m; i++) {
    for (int j = 0; j < k; j++) {
      A[i+m*j] = 1.0;
      std::cout << A[i+m*j] << " ";
    }
    std::cout << std::endl;
  }

  for(int i = 0; i < k; i++) {
    for (int j = 0; j < n; j++) {
      B[i+j*k] = 1.0;
      std::cout << B[i+j*k] << " ";
    }
    std::cout << std::endl;
  }
  cuMultiplyMat(m, n, k, A, B, C);
  
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      std::cout << C[i+m*j] << " ";
    }
    std::cout << std::endl;
  }
  return 0;
}
*/
