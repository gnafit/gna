#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <typeinfo>
#include <iostream>
#include <hip/hip_runtime.h>
#include "GNAcuMath.h"

/**
  *  Generation of Identity matrix on GPU memory
  *  TODO: find an optimal grid and block sizes!
  */
__global__ void GenIdentity (int n, double * mat) {
  int x = blockDim.x*blockIdx.x + threadIdx.x;
  int y = blockDim.y*blockIdx.y + threadIdx.y;
  if (x < n && y < n)
  mat[x + n * y] = (x == y) ? 1.0 : 0.0; 
}

void cuInverseMat(int matSize, double* InMat, double* OutMat) {
  hipSetDevice(0);
  hipblasHandle_t handle;
  hipblasStatus_t ret;
  hipError_t err;
  
  hipStream_t stream1, stream2;
  hipStreamCreate ( &stream1);
  hipStreamCreate ( &stream2);

  ret = hipblasCreate(&handle);
  if(ret!=HIPBLAS_STATUS_SUCCESS){
    exit(EXIT_FAILURE);
  }

  double* devInMat;
  double* devOutMat;
  hipMalloc((void**)&devInMat,  matSize*matSize*sizeof(double));
  hipMalloc((void**)&devOutMat,  matSize*matSize*sizeof(double));

  GenIdentity<<<dim3(matSize/32 + 1, matSize/32 + 1), dim3(32,32), 0, stream2>>>(matSize, devOutMat);
  //hipDeviceSynchronize();

  err = hipMemcpyAsync(devInMat, InMat, matSize*matSize*sizeof(double), hipMemcpyHostToDevice, stream1);
  if(err!=hipSuccess) {
    exit(EXIT_FAILURE);
  }

  double alpha = 1.0;
/**
  *  Solve A*x = alpha * B to invert matrix. In this case B is Identity, alpha == 1.
  */
  ret = hipblasDtrsm(handle,
          HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,
          matSize, matSize, &alpha, devInMat, matSize, devOutMat, matSize);
  hipDeviceSynchronize();
  if(ret!=HIPBLAS_STATUS_SUCCESS) {
    exit(EXIT_FAILURE);
  }

  err = hipMemcpyAsync(OutMat, devOutMat, matSize*matSize*sizeof(double), hipMemcpyDeviceToHost, stream1);

  if(err!=hipSuccess) {
    exit(EXIT_FAILURE);
  }

  hipFree(devInMat);
  hipFree(devOutMat);
}

/*int main () {
  int n = 200;
  double* inM = new double[n*n];
  double* outM = new double[n*n];
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      if (i == j) {
        inM[i + n*j] = i + j + 1;
        outM[i + n*j] = 1;
      }
      else {
        inM[i + n*j] = 0;
        outM[i + n*j] = 0;
      }
    }
  }

  for(int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      if (outM[i + j*n] != 0)
      std::cout << inM[i + j*n] << " ";
    }
  //  std::cout << std::endl;
  }

  cuInverseMat(n, inM,  outM);

  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      if (outM[i + j*n] != 0) 
      std::cout << outM[i + j*n] << " ";
    }
    std::cout << std::endl;
  }

  return 0;
}
*/
