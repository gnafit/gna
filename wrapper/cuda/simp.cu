#include "hip/hip_runtime.h"
#include <cstdio>
#include <math.h>
#include <hip/hip_runtime.h>
#include <>
#include "simp.h"

const int N = 50;

__global__ void  add( int *a,int *b, int *c ) {
int tid = blockIdx.x;    
if (tid < N)
        c[tid] = a[tid] + b[tid];
}

void simpfun() {
    int  a[N], b[N], c[N];
    int  *dev_a, *dev_b, *dev_c;
    hipSetDevice(0);
    hipMalloc( (void**)&dev_a, N * sizeof(int ) );
    hipMalloc( (void**)&dev_b, N * sizeof(int) );
    hipMalloc( (void**)&dev_c, N * sizeof(int) );
    
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice );
    
    add<<<N,1>>>( dev_a, dev_b, dev_c ); 
    
    hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost );
    for (int i=0; i<N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i] );
    }
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );
}
